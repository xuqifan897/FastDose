#include <boost/filesystem.hpp>
#include "hip/hip_runtime.h"
#include "fastdose.cuh"
#include "IMRTArgs.h"
#include "IMRTInit.cuh"
#include "IMRTBeamBundle.cuh"
#include "IMRTDoseMat.cuh"
#include "IMRTDoseMatEns.cuh"
#include "IMRTDoseMatEigen.cuh"
#include "IMRTDebug.cuh"

namespace fs = boost::filesystem;
namespace fd = fastdose;

int main(int argc, char** argv) {
    if (IMRT::argparse(argc, argv))
        return 0;

    int mode = IMRT::getarg<int>("mode");
    int deviceIdx = IMRT::getarg<int>("deviceIdx");
    hipSetDevice(deviceIdx);

    if (fd::showDeviceProperties(deviceIdx)) {
        std::cerr << "Cannot show device properties." << std::endl;
        return 1;
    }

    IMRT::Params params;
    if (IMRT::ParamsInit(params)) {
        std::cerr << "Paramsters initialization error." << std::endl;
        return 1;
    }

    std::vector<IMRT::StructInfo> structs;
    if (IMRT::StructsInit(structs)) {
        std::cerr << "Structure initialization error." << std::endl;
        return 1;
    }

    IMRT::MatCSREnsemble* matEns = nullptr;
    if (mode == 0 || mode == 1) {
        // for mode 0, do dose calculation and store the result.
        // for mode 1, do dose calculation and perform beam orientation optimization
        fd::DENSITY_h density_h;
        fd::DENSITY_d density_d;
        if (IMRT::densityInit(density_h, density_d, structs)) {
            std::cerr << "Density initialization error." << std::endl;
            return 1;
        }

        fd::SPECTRUM_h spectrum_h;
        if (IMRT::specInit(spectrum_h)) {
            std::cerr << "Spectrum initialization error." << std::endl;
            return 1;
        }

        fd::KERNEL_h kernel_h;
        if (IMRT::kernelInit(kernel_h)) {
            std::cerr << "Kernel initialization error." << std::endl;
            return 1;
        }

        std::vector<IMRT::BeamBundle> beam_bundles;
        if (IMRT::BeamBundleInit(beam_bundles, density_h, structs)) {
            std::cerr << "Beam bundles initialization error." << std::endl;
            return 1;
        }

        if (IMRT::DoseMatConstruction(beam_bundles, density_d, spectrum_h, kernel_h, &matEns)) {
            std::cerr << "Dose matrix construction error." << std::endl;
            return 1;
        }

        if (mode == 0) {
            fs::path doseMatFolder(IMRT::getarg<std::string>("outputFolder"));
            doseMatFolder /= std::string("doseMatFolder");
            matEns->tofile(doseMatFolder.string());
            fs::path fluenceMapPath = doseMatFolder / std::string("fluenceMap.bin");
            IMRT::beamletFlagSave(beam_bundles, fluenceMapPath.string());
            return 0;
        }
        return 0; // ignore mode 1 at this time.
    }

    IMRT::MatCSR64 SpOARmat, SpOARmatT;
    IMRT::MatCSR64 SpFluenceGrad, SpFluenceGradT;
    if (mode == 2) {
        fs::path doseMatFolder(IMRT::getarg<std::string>("outputFolder"));
        doseMatFolder /= std::string("doseMatFolder");
        IMRT::OARFiltering(doseMatFolder.string(), structs, SpOARmat, SpOARmatT);

        int fluenceDim = IMRT::getarg<int>("fluenceDim");
        fs::path fluenceMapPath = doseMatFolder / std::string("fluenceMap.bin");
        IMRT::fluenceGradInit(SpFluenceGrad, SpFluenceGradT, fluenceMapPath.string(), fluenceDim);
    }
}