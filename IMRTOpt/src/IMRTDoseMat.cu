#include "hip/hip_runtime.h"
#include <fstream>
#include <iomanip>
#include <boost/filesystem.hpp>
namespace fs = boost::filesystem;

#include "IMRTDoseMat.cuh"
#include "IMRTArgs.h"
#include "IMRTgeom.cuh"

namespace fd = fastdose;

bool IMRT::MatCSR::dense2sparse(
    float* d_dense, size_t num_rows, size_t num_cols, size_t ld
) {
    checkCudaErrors(hipMalloc((void**)(&this->d_csr_offsets),
        (num_rows + 1) * sizeof(int)));
    
    hipsparseHandle_t handle = nullptr;
    hipsparseDnMatDescr_t matDense;
    void* dBufferConstruct = nullptr;
    size_t bufferSize = 0;

    checkCusparse(hipsparseCreate(&handle))

    checkCusparse(hipsparseCreateDnMat(
        &matDense, num_rows, num_cols, ld,
        d_dense, HIP_R_32F, HIPSPARSE_ORDER_ROW))

    checkCusparse(hipsparseCreateCsr(
        &(this->matA), num_rows, num_cols, 0,
        d_csr_offsets, nullptr, nullptr,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F))
    
    // allocate an external buffer if needed
    checkCusparse(hipsparseDenseToSparse_bufferSize(
        handle, matDense, this->matA,
        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
        &bufferSize))
    checkCudaErrors(hipMalloc((void**) &dBufferConstruct, bufferSize));
    
    // execute Dense to Sparse conversion
    checkCusparse(hipsparseDenseToSparse_analysis(
        handle, matDense, this->matA,
        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBufferConstruct))

    // get the number of non-zero elements
    int64_t num_rows_tmp, num_cols_tmp;
    checkCusparse(hipsparseSpMatGetSize(
        this->matA, &num_rows_tmp, &num_cols_tmp, &(this->nnz)))
    
    // allocate CSR column indices and values
    checkCudaErrors(hipMalloc((void**) &(this->d_csr_columns), nnz*sizeof(int)));
    checkCudaErrors(hipMalloc((void**) &(this->d_csr_values), nnz*sizeof(float)));
    // reset offsets, column indices, and values pointers
    checkCusparse(hipsparseCsrSetPointers(this->matA,
        this->d_csr_offsets, this->d_csr_columns, this->d_csr_values))
    
    // execute Dense to Sparse conversion
    checkCusparse(hipsparseDenseToSparse_convert(handle, matDense, this->matA,
        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBufferConstruct))
    
    checkCudaErrors(hipFree(dBufferConstruct));
    checkCusparse(hipsparseDestroyDnMat(matDense))
    checkCusparse(hipsparseDestroy(handle))
    return 0;
}

bool IMRT::DoseMatConstruction(
    std::vector<BeamBundle>& beam_bundles,
    fd::DENSITY_d& density_d,
    fd::SPECTRUM_h& spectrum_h,
    fd::KERNEL_h& kernel_h,
    hipStream_t stream
) {
    // prepare the sparse dose matrices
    std::vector<MatCSR> SparseMatArray;

    hipEvent_t start, stop, globalStart, globalStop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventCreate(&globalStart));
    checkCudaErrors(hipEventCreate(&globalStop));
    float milliseconds;

    // firstly, calculate Terma and Dose
    int concurrency = getarg<int>("concurrency");
    float extent = getarg<float>("extent");
    int iterations = (beam_bundles.size() + concurrency - 1) / concurrency;
    SparseMatArray.resize(iterations);

    hipEventRecord(globalStart);
    for (int i=0; i<iterations; i++) {
        hipEventRecord(start);

        std::vector<fd::BEAM_d> beamlets;
        int beam_bundle_idx_begin = i * concurrency;
        int beam_bundle_idx_end = (i + 1) * concurrency;
        beam_bundle_idx_end = min(beam_bundle_idx_end, (int)(beam_bundles.size()));

        int nBeamlets = 0;
        for (int j=beam_bundle_idx_begin; j<beam_bundle_idx_end; j++) {
            nBeamlets += beam_bundles[j].beams_h.size();
        }
        beamlets.resize(nBeamlets);
        int count = 0;
        for (int j=beam_bundle_idx_begin; j<beam_bundle_idx_end; j++) {
            auto& current = beam_bundles[j];
            for (int k=0; k<current.beams_h.size(); k++) {
                fd::beam_h2d(current.beams_h[k], beamlets[count]);
                count ++;
            }
        }

        // preparation
        std::vector<fd::d_BEAM_d> h_beams;
        h_beams.reserve(nBeamlets);
        for (int j=0; j<nBeamlets; j++) {
            h_beams.push_back(fd::d_BEAM_d(beamlets[j]));
        }
        fd::d_BEAM_d* d_beams = nullptr;
        checkCudaErrors(hipMalloc((void**)(&d_beams), nBeamlets*sizeof(fd::d_BEAM_d)));
        checkCudaErrors(hipMemcpy(d_beams, h_beams.data(),
            nBeamlets*sizeof(fd::d_BEAM_d), hipMemcpyHostToDevice));

        // allocate fluence array
        std::vector<float*> h_fluence_array(nBeamlets, nullptr);
        for (int j=0; j<nBeamlets; j++)
            h_fluence_array[j] = beamlets[j].fluence;
        float** d_fluence_array = nullptr;
        checkCudaErrors(hipMalloc((void***)(&d_fluence_array), nBeamlets*sizeof(float*)));
        checkCudaErrors(hipMemcpy(d_fluence_array, h_fluence_array.data(),
            nBeamlets*sizeof(float*), hipMemcpyHostToDevice));

        // allocate Terma_array
        std::vector<float*> h_TermaBEV_array(nBeamlets, nullptr);
        for (int j=0; j<nBeamlets; j++)
            h_TermaBEV_array[j] = beamlets[j].TermaBEV;
        float** d_TermaBEV_array = nullptr;
        checkCudaErrors(hipMalloc((void***)(&d_TermaBEV_array), nBeamlets*sizeof(float*)));
        checkCudaErrors(hipMemcpy(d_TermaBEV_array, h_TermaBEV_array.data(),
            nBeamlets*sizeof(float*), hipMemcpyHostToDevice));

        // allocate DenseBEV_array
        std::vector<float*> h_DensityBEV_array(nBeamlets, nullptr);
        for (int j=0; j<nBeamlets; j++)
            h_DensityBEV_array[j] = beamlets[j].DensityBEV;
        float** d_DensityBEV_array = nullptr;
        checkCudaErrors(hipMalloc((void***)(&d_DensityBEV_array), nBeamlets*sizeof(float*)));
        checkCudaErrors(hipMemcpy(d_DensityBEV_array, h_DensityBEV_array.data(),
            nBeamlets*sizeof(float*), hipMemcpyHostToDevice));
        
        // allocate DoseBEV_array
        std::vector<float*> h_DoseBEV_array(nBeamlets, nullptr);
        for (int j=0; j<nBeamlets; j++)
            h_DoseBEV_array[j] = beamlets[j].DoseBEV;
        float** d_DoseBEV_array = nullptr;
        checkCudaErrors(hipMalloc((void***)(&d_DoseBEV_array), nBeamlets*sizeof(float*)));
        checkCudaErrors(hipMemcpy(d_DoseBEV_array, h_DoseBEV_array.data(),
            nBeamlets*sizeof(float*), hipMemcpyHostToDevice));

        size_t fmap_npixels = beamlets[0].fmap_size.x * beamlets[0].fmap_size.y;

        fd::TermaComputeCollective(
            fmap_npixels,
            nBeamlets,
            d_beams,
            d_fluence_array,
            d_TermaBEV_array,
            d_DensityBEV_array,
            density_d,
            spectrum_h,
            stream
        );

        fd::DoseComputeCollective(
            fmap_npixels,
            nBeamlets,
            d_beams,
            d_TermaBEV_array,
            d_DensityBEV_array,
            d_DoseBEV_array,
            kernel_h.nTheta,
            kernel_h.nPhi,
            stream
        );

        // clean up
        checkCudaErrors(hipFree(d_DoseBEV_array));
        checkCudaErrors(hipFree(d_DensityBEV_array));
        checkCudaErrors(hipFree(d_TermaBEV_array));
        checkCudaErrors(hipFree(d_fluence_array));
        
        float* d_dense_dose;
        size_t denseDoseSize = density_d.VolumeDim.x
            * density_d.VolumeDim.y * density_d.VolumeDim.z
            * beamlets.size();
        checkCudaErrors(hipMalloc((void**)&d_dense_dose, denseDoseSize*sizeof(float)));
        checkCudaErrors(hipMemset(d_dense_dose, 0, denseDoseSize*sizeof(float)));
        BEV2PVCSInterp(&d_dense_dose, beamlets, d_beams, density_d, 5, extent, stream);
        checkCudaErrors(hipFree(d_beams));

        MatCSR& currentMat = SparseMatArray[i];
        size_t nColumns = density_d.VolumeDim.x * density_d.VolumeDim.y * density_d.VolumeDim.z;
        currentMat.dense2sparse(d_dense_dose, beamlets.size(), nColumns, nColumns);
        checkCudaErrors(hipFree(d_dense_dose));

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);

        size_t free_bytes, total_bytes;
        checkCudaErrors(hipMemGetInfo(&free_bytes, &total_bytes));
        float free_GB = (float)free_bytes * (1.0f / 1024) * (1.0f / 1024) * (1.0f / 1024);
        float total_GB = (float)total_bytes * (1.0f / 1024) * (1.0f / 1024) * (1.0f / 1024);
        std::cout << std::fixed << std::setprecision(2)
            << "Beams " << beam_bundle_idx_begin + 1 << " ~ " << beam_bundle_idx_end << " / " << beam_bundles.size()
            << ", number of beamlets: " << nBeamlets << ", time elapsed: " << milliseconds
            << "[ms], free memory: " << free_GB << " / " << total_GB << " GB" << std::endl;
    }
    hipEventRecord(globalStop);
    hipEventSynchronize(globalStop);
    hipEventElapsedTime(&milliseconds, globalStart, globalStop);
    std::cout << std::endl << "Concurrency: " << concurrency
        << ", total dose calculation time: "
        << milliseconds / 1000 << "s." << std::endl;
    return 0;
}