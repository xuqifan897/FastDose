#include "hip/hip_runtime.h"
#include "IMRTDoseMat.cuh"
#include "IMRTArgs.h"
#include "IMRTgeom.cuh"

namespace fd = fastdose;

bool IMRT::MatCSR::dense2sparse(
    float* d_dense, int num_rows, int num_cols, int ld
) {
    checkCudaErrors(hipMalloc((void**)(&this->d_csr_offsets),
        (num_rows + 1) * sizeof(int)));
    
    hipsparseHandle_t handle = nullptr;
    hipsparseDnMatDescr_t matDense;
    void* dBufferConstruct = nullptr;
    size_t bufferSize = 0;

    checkCusparse(hipsparseCreate(&handle))

    checkCusparse(hipsparseCreateDnMat(
        &matDense, num_rows, num_cols, ld,
        d_dense, HIP_R_32F, HIPSPARSE_ORDER_ROW))

    checkCusparse(hipsparseCreateCsr(
        &(this->matA), num_rows, num_cols, 0,
        d_csr_offsets, nullptr, nullptr,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F))
    
    // allocate an external buffer if needed
    checkCusparse(hipsparseDenseToSparse_bufferSize(
        handle, matDense, this->matA,
        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
        &bufferSize))
    checkCudaErrors(hipMalloc((void**) &dBufferConstruct, bufferSize));
    
    // execute Sparse to Dense conversion
    checkCusparse(hipsparseDenseToSparse_analysis(
        handle, matDense, this->matA,
        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBufferConstruct))

    // get the number of non-zero elements
    int64_t num_rows_tmp, num_cols_tmp;
    checkCusparse(hipsparseSpMatGetSize(
        this->matA, &num_rows_tmp, &num_cols_tmp, &(this->nnz)))
    
    // allocate CSR column indices and values
    checkCudaErrors(hipMalloc((void**) &(this->d_csr_columns), nnz*sizeof(int)));
    checkCudaErrors(hipMalloc((void**) &(this->d_csr_values), nnz*sizeof(float)));
    // reset offsets, column indices, and values pointers
    checkCusparse(hipsparseCsrSetPointers(this->matA,
        this->d_csr_offsets, this->d_csr_columns, this->d_csr_values))
    
    // execute Sparse to Dense conversion
    checkCusparse(hipsparseDenseToSparse_convert(handle, matDense, this->matA,
        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBufferConstruct))
    
    checkCudaErrors(hipFree(dBufferConstruct));
    checkCusparse(hipsparseDestroyDnMat(matDense))
    checkCusparse(hipsparseDestroy(handle))
    return 0;
}

bool IMRT::DoseMatConstruction(
    std::vector<BeamBundle>& beam_bundles,
    fd::DENSITY_d& density_d,
    fd::SPECTRUM_h& spectrum_h,
    fd::KERNEL_h& kernel_h,
    hipStream_t stream
) {
    // prepare the sparse dose matrices
    std::vector<MatCSR> SparseMatArray;

    // firstly, calculate Terma and Dose
    int concurrency = getarg<int>("concurrency");
    float extent = getarg<float>("extent");
    int iterations = (beam_bundles.size() + concurrency - 1) / concurrency;
    for (int i=0; i<iterations; i++) {
        std::vector<fd::BEAM_d> beamlets;
        int beam_bundle_idx_begin = i * concurrency;
        int beam_bundle_idx_end = (i + 1) * concurrency;
        beam_bundle_idx_end = min(beam_bundle_idx_end, (int)(beam_bundles.size()));

        int nBeamlets = 0;
        for (int j=beam_bundle_idx_begin; j<beam_bundle_idx_end; j++) {
            nBeamlets += beam_bundles[j].beams_h.size();
        }
        beamlets.resize(nBeamlets);
        int count = 0;
        for (int j=beam_bundle_idx_begin; j<beam_bundle_idx_end; j++) {
            auto& current = beam_bundles[j];
            for (int k=0; k<current.beams_h.size(); k++) {
                fd::beam_h2d(current.beams_h[k], beamlets[count]);
                count ++;
            }
        }

        // preparation
        std::vector<fd::d_BEAM_d> h_beams;
        h_beams.reserve(nBeamlets);
        for (int j=0; j<nBeamlets; j++) {
            h_beams.push_back(fd::d_BEAM_d(beamlets[j]));
        }
        fd::d_BEAM_d* d_beams = nullptr;
        checkCudaErrors(hipMalloc((void**)(&d_beams), nBeamlets*sizeof(fd::d_BEAM_d)));
        checkCudaErrors(hipMemcpy(d_beams, h_beams.data(),
            nBeamlets*sizeof(fd::d_BEAM_d), hipMemcpyHostToDevice));

        // allocate fluence array
        std::vector<float*> h_fluence_array(nBeamlets, nullptr);
        for (int j=0; j<nBeamlets; j++)
            h_fluence_array[j] = beamlets[j].fluence;
        float** d_fluence_array = nullptr;
        checkCudaErrors(hipMalloc((void***)(&d_fluence_array), nBeamlets*sizeof(float*)));
        checkCudaErrors(hipMemcpy(d_fluence_array, h_fluence_array.data(),
            nBeamlets*sizeof(float*), hipMemcpyHostToDevice));

        // allocate Terma_array
        std::vector<float*> h_TermaBEV_array(nBeamlets, nullptr);
        for (int j=0; j<nBeamlets; j++)
            h_TermaBEV_array[j] = beamlets[j].TermaBEV;
        float** d_TermaBEV_array = nullptr;
        checkCudaErrors(hipMalloc((void***)(&d_TermaBEV_array), nBeamlets*sizeof(float*)));
        checkCudaErrors(hipMemcpy(d_TermaBEV_array, h_TermaBEV_array.data(),
            nBeamlets*sizeof(float*), hipMemcpyHostToDevice));

        // allocate DenseBEV_array
        std::vector<float*> h_DensityBEV_array(nBeamlets, nullptr);
        for (int j=0; j<nBeamlets; j++)
            h_DensityBEV_array[j] = beamlets[j].DensityBEV;
        float** d_DensityBEV_array = nullptr;
        checkCudaErrors(hipMalloc((void***)(&d_DensityBEV_array), nBeamlets*sizeof(float*)));
        checkCudaErrors(hipMemcpy(d_DensityBEV_array, h_DensityBEV_array.data(),
            nBeamlets*sizeof(float*), hipMemcpyHostToDevice));
        
        // allocate DoseBEV_array
        std::vector<float*> h_DoseBEV_array(nBeamlets, nullptr);
        for (int j=0; j<nBeamlets; j++)
            h_DoseBEV_array[j] = beamlets[j].DoseBEV;
        float** d_DoseBEV_array = nullptr;
        checkCudaErrors(hipMalloc((void***)(&d_DoseBEV_array), nBeamlets*sizeof(float*)));
        checkCudaErrors(hipMemcpy(d_DoseBEV_array, h_DoseBEV_array.data(),
            nBeamlets*sizeof(float*), hipMemcpyHostToDevice));

        size_t fmap_npixels = beamlets[0].fmap_size.x * beamlets[0].fmap_size.y;

        #if TIMING
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);
        #endif

        fd::TermaComputeCollective(
            fmap_npixels,
            nBeamlets,
            d_beams,
            d_fluence_array,
            d_TermaBEV_array,
            d_DensityBEV_array,
            density_d,
            spectrum_h,
            stream
        );

        #if TIMING
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float milliseconds = 0.0f;
            hipEventElapsedTime(&milliseconds, start, stop);
            std::cout << "Terma time elapsed: " << milliseconds << " [ms]" << std::endl;
        #endif

        #if TIMING
            hipEventRecord(start);
        #endif

        fd::DoseComputeCollective(
            fmap_npixels,
            nBeamlets,
            d_beams,
            d_TermaBEV_array,
            d_DensityBEV_array,
            d_DoseBEV_array,
            kernel_h.nTheta,
            kernel_h.nPhi,
            stream
        );

        #if TIMING
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&milliseconds, start, stop);
            std::cout << "Dose time elapsed: " << milliseconds << " [ms]" << std::endl;
        #endif

        // clean up
        checkCudaErrors(hipFree(d_DoseBEV_array));
        checkCudaErrors(hipFree(d_DensityBEV_array));
        checkCudaErrors(hipFree(d_TermaBEV_array));
        checkCudaErrors(hipFree(d_fluence_array));
        
        float* d_dense_dose;
        size_t denseDoseSize = density_d.VolumeDim.x
            * density_d.VolumeDim.y * density_d.VolumeDim.z
            * beamlets.size();
        checkCudaErrors(hipMalloc((void**)&d_dense_dose, denseDoseSize*sizeof(float)));
        checkCudaErrors(hipMemset(d_dense_dose, 0, denseDoseSize*sizeof(float)));
        BEV2PVCSInterp(&d_dense_dose, beamlets, d_beams, density_d, 5, extent, stream);

        checkCudaErrors(hipFree(d_beams));

        #if false
            size_t freeBytes, totalBytes;
            hipMemGetInfo(&freeBytes, &totalBytes);
            std::cout << "Free memory: " << (float)freeBytes / (1<<30) << "GB \n"
                "Total memory: " << (float)totalBytes / (1<<30) << "GB." << std::endl;
        #endif

        #include "IMRTDirectInterp.cpp.in"

        // for debug purposes
        break;
    }
    return 0;
}