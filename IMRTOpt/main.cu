#include <boost/filesystem.hpp>
#include "hip/hip_runtime.h"
#include "fastdose.cuh"
#include "IMRTArgs.h"
#include "IMRTInit.cuh"
#include "IMRTBeamBundle.cuh"
#include "IMRTDoseMat.cuh"
#include "IMRTDebug.cuh"

namespace fs = boost::filesystem;
namespace fd = fastdose;

int main(int argc, char** argv) {
    if (IMRT::argparse(argc, argv))
        return 0;

    int mode = IMRT::getarg<int>("mode");
    int deviceIdx = IMRT::getarg<int>("deviceIdx");
    hipSetDevice(deviceIdx);

    if (fd::showDeviceProperties(deviceIdx)) {
        std::cerr << "Cannot show device properties." << std::endl;
        return 1;
    }

    std::vector<IMRT::StructInfo> structs;
    if (IMRT::StructsInit(structs)) {
        std::cerr << "Structure initialization error." << std::endl;
        return 1;
    }

    IMRT::MatCSREnsemble* matEns = nullptr;
    if (mode == 0 || mode == 1) {
        // for mode 0, do dose calculation and store the result.
        // for mode 1, do dose calculation and perform beam orientation optimization
        fd::DENSITY_h density_h;
        fd::DENSITY_d density_d;
        if (IMRT::densityInit(density_h, density_d, structs)) {
            std::cerr << "Density initialization error." << std::endl;
            return 1;
        }

        fd::SPECTRUM_h spectrum_h;
        if (IMRT::specInit(spectrum_h)) {
            std::cerr << "Spectrum initialization error." << std::endl;
            return 1;
        }

        fd::KERNEL_h kernel_h;
        if (IMRT::kernelInit(kernel_h)) {
            std::cerr << "Kernel initialization error." << std::endl;
            return 1;
        }

        std::vector<IMRT::BeamBundle> beam_bundles;
        if (IMRT::BeamBundleInit(beam_bundles, density_h, structs)) {
            std::cerr << "Beam bundles initialization error." << std::endl;
            return 1;
        }

        if (IMRT::DoseMatConstruction(beam_bundles, density_d, spectrum_h, kernel_h, &matEns)) {
            std::cerr << "Dose matrix construction error." << std::endl;
            return 1;
        }
    }
    
    fs::path doseMatFolder(IMRT::getarg<std::string>("outputFolder"));
    doseMatFolder /= std::string("doseMatFolder");
    if (mode == 0) {
        matEns->tofile(doseMatFolder.string());
        return ;
    } else if (mode == 2) {
        const std::vector<int>& phantomDim = IMRT::getarg<std::vector<int>>("phantomDim");
        size_t numColsPerMat = phantomDim[0] * phantomDim[1] * phantomDim[2];
        matEns = new IMRT::MatCSREnsemble(numColsPerMat);
        matEns->fromfile(doseMatFolder.string());
    }

    #if false
        IMRT::sparseValidation(matEns);
    #endif

    IMRT::MatCSR matrix;
    matrix.fuseEnsemble(*matEns);
}