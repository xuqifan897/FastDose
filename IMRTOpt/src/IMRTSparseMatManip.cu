#include "IMRTDoseMat.cuh"
#include <iostream>
#include <chrono>
#include <boost/filesystem.hpp>
namespace fs = boost::filesystem;


bool IMRT::MatCSR32_fromfile(const std::string resultFolder,
    size_t numColsPerMat, const std::vector<StructInfo>& structs
) {
    #if TIMING
        auto time0 = std::chrono::high_resolution_clock::now();
    #endif

    std::vector<std::pair<std::vector<size_t>, std::string>> inputQueue {
        {std::vector<size_t>(), std::string("NonZeroElements")},
        {std::vector<size_t>(), std::string("numRowsPerMat")}
    };

    size_t numMatrices = 0;
    for (auto& a: inputQueue) {
        std::vector<size_t>& array = a.first;
        const std::string& name = a.second;
        fs::path fullFile = fs::path(resultFolder) / (name + std::string(".bin"));
        std::ifstream f(fullFile.string());
        if (! f.is_open()) {
            std::cerr << "Cannot open file: " << fullFile << std::endl;
            return 1;
        }
        if (numMatrices == 0) {
            f.seekg(0, std::ios::end);
            numMatrices = f.tellg() / sizeof(size_t);
            f.seekg(0, std::ios::beg);
        }
        array.resize(numMatrices);
        f.read((char*)array.data(), numMatrices*sizeof(size_t));
        f.close();
    }

    // fill offsetsBuffer
    fs::path offsetsBufferFile = fs::path(resultFolder) / std::string("offsetsBuffer.bin");
    std::ifstream f(offsetsBufferFile.string());
    if (! f.is_open()) {
        std::cerr << "Cannot open file: " << offsetsBufferFile << std::endl;
        return 1;
    }
    f.seekg(0, std::ios::end);
    size_t offsetsBufferSize = f.tellg() / sizeof(size_t);
    f.seekg(0, std::ios::beg);
    std::vector<size_t> h_offsetsBuffer(offsetsBufferSize, 0);
    f.read((char*)(h_offsetsBuffer.data()), offsetsBufferSize*sizeof(size_t));
    f.close();

    size_t nnz = 0;
    fs::path columnsBufferFile = fs::path(resultFolder) / std::string("columnsBuffer.bin");
    f.open(columnsBufferFile.string());
    if (! f.is_open()) {
        std::cerr << "Cannot open file: " << columnsBufferFile << std::endl;
        return 1;
    }
    f.seekg(0, std::ios::end);
    nnz = f.tellg() / sizeof(size_t);
    f.seekg(0, std::ios::beg);
    std::vector<size_t> h_columnsBuffer(nnz, 0);
    f.read((char*)(h_columnsBuffer.data()), nnz*sizeof(size_t));
    f.close();

    fs::path valuesBufferFile = fs::path(resultFolder) / std::string("valuesBuffer.bin");
    f.open(valuesBufferFile.string());
    if (! f.is_open()) {
        std::cerr << "Cannot open file: " << valuesBufferFile << std::endl;
        return 1;
    }
    std::vector<float> m_values(nnz, 0);
    f.read((char*)m_values.data(), nnz*sizeof(float));
    f.close();

    #if TIMING
        auto time1 = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(time1 - time0);
        std::cout << "Time elapsed to load data from file: " << std::fixed
            << duration.count() * 0.001f << " [s]" << std::endl;
    #endif

    // convert the size_t array to int array in parallel
    std::vector<int> m_offsets(offsetsBufferSize, 0);
    std::vector<int> m_columns(nnz, 0);
    for (size_t i=0; i<offsetsBufferSize; i++)
        m_offsets[i] = static_cast<int>(h_offsetsBuffer[i]);
    for (size_t i=0; i<nnz; i++) {
        m_columns[i] = static_cast<int>(h_columnsBuffer[i]);
    }

    #if TIMING
        auto time2 = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(time2 - time1);
        std::cout << "Time elapsed to convert size_t to int: "
            << duration.count() * 0.001f << " [s]" << std::endl;
    #endif


    // load the data to device
    int* d_offsets = nullptr;
    int* d_columns = nullptr;
    float* d_values = nullptr;
    checkCudaErrors(hipMalloc((void**)&d_offsets, offsetsBufferSize*sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&d_columns, nnz*sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&d_values, nnz*sizeof(float)));
    checkCudaErrors(hipMemcpy(d_offsets, h_offsetsBuffer.data(),
        offsetsBufferSize*sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_columns, m_columns.data(),
        nnz*sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_values, m_values.data(),
        nnz*sizeof(float), hipMemcpyHostToDevice));

    #if TIMING
        auto time3 = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(time3 - time2);
        std::cout << "Time elapsed to copy host data to device: "
            << duration.count() * 0.001f << " [s]" << std::endl;
    #endif

    // create matrices
    std::vector<MatCSR32> matricesT(numMatrices);
    size_t cumuRows = 0;
    size_t cumuNnz = 0;
    for (int i=0; i<numMatrices; i++) {
        // set the pointers
        int* d_currentOffsets = d_offsets + cumuRows;
        int* d_currentColumns = d_columns + cumuNnz;
        float* d_currentValues = d_values + cumuNnz;

        size_t currentNumRows = inputQueue[1].first[i];
        size_t currentNnz = inputQueue[0].first[i];
        cumuRows += currentNumRows + 1;
        cumuNnz += currentNnz;

        MatCSR32& currentMat = matricesT[i];
        currentMat.d_csr_offsets = d_currentOffsets;
        currentMat.d_csr_columns = d_currentColumns;
        currentMat.d_csr_values = d_currentValues;
        currentMat.numRows = static_cast<int>(currentNumRows);
        currentMat.numCols = static_cast<int>(numColsPerMat);
        currentMat.nnz = static_cast<int64_t>(currentNnz);
        checkCusparse(hipsparseCreateCsr(
            &currentMat.matA, currentNumRows, numColsPerMat, currentNnz,
            d_currentOffsets, d_currentColumns, d_currentValues,
            HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    }
    #if TIMING
        auto time4 = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(time4 - time3);
        std::cout << "Time elapsed to construct the sparse matrices: "
            << duration.count() * 0.001f << " [s]" << std::endl;
    #endif

    // construct OAR filtering matrix
    MatCSR32 matFilter, matFilterT;
    getOARFilter(matFilter, matFilterT, structs, numColsPerMat);
    #if TIMING
        auto time5 = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(time5 - time4);
        std::cout << "Time elapsed to construct the OAR filtering matrices: "
            << duration.count() * 0.001f << " [s]" << std::endl;
    #endif

    #if false
        test_MatFilter(matFilter, matFilterT);
    #endif

    std::vector<MatCSR32> OARMatricesT(numMatrices);
    int* d_bufferOffsets = nullptr;
    int* d_bufferColumns = nullptr;
    float* d_bufferValues = nullptr;
    OARFiltering(OARMatricesT, matricesT, matFilter, matFilterT,
        &d_bufferOffsets, &d_bufferColumns, &d_bufferValues);
    #if TIMING
        auto time6 = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::milliseconds>(time6 - time5);
        std::cout << "Time elapsed to construct the dose loading matrix "
            "restricted to OAR volumes: " << duration.count() * 0.001f
            << " [s]" << std::endl; 
    #endif

    // clean up
    for (int i=0; i<numMatrices; i++) {
        MatCSR32& currentMatOAR = OARMatricesT[i];
        checkCusparse(hipsparseDestroySpMat(currentMatOAR.matA));
        currentMatOAR.matA = nullptr;
        currentMatOAR.d_csr_offsets = nullptr;
        currentMatOAR.d_csr_columns = nullptr;
        currentMatOAR.d_csr_values = nullptr;

        MatCSR32& currentMat = matricesT[i];
        checkCusparse(hipsparseDestroySpMat(currentMat.matA));
        currentMat.matA = nullptr;
        currentMat.d_csr_offsets = nullptr;
        currentMat.d_csr_columns = nullptr;
        currentMat.d_csr_values = nullptr;
    }
    checkCudaErrors(hipFree(d_bufferOffsets));
    checkCudaErrors(hipFree(d_bufferColumns));
    checkCudaErrors(hipFree(d_bufferValues));
    checkCudaErrors(hipFree(d_offsets));
    checkCudaErrors(hipFree(d_columns));
    checkCudaErrors(hipFree(d_values));
    return 0;
}


bool IMRT::getOARFilter(MatCSR32& matFilter, MatCSR32& matFilterT,
    const std::vector<StructInfo>& structs, size_t nVoxels
) {
    std::vector<int> nonZeroVoxels;
    int totalCount = 0;
    for (int i=0; i<structs.size(); i++) {
        const StructInfo& currentStruct = structs[i];
        if (currentStruct.maxWeights < eps_fastdose &&
            currentStruct.minDoseTargetWeights < eps_fastdose &&
            currentStruct.OARWeights < eps_fastdose) {
            std::cout << "Structure: " << currentStruct.name
                << " is irrelevant in the optimization, skip." << std::endl;
            continue;
        }

        int localCount = 0;
        if (nVoxels != currentStruct.size.x
            * currentStruct.size.y * currentStruct.size.z
        ) {
            std::cerr << "Number of voxels not consistent across structures." << std::endl;
            return 1;
        }
        for (int j=0; j<nVoxels; j++)
            localCount += (currentStruct.mask[j] > 0);
        nonZeroVoxels.push_back(localCount);
        totalCount += localCount;
        std::cout << "Structure: " << currentStruct.name
            << ", non-zero voxels: " << localCount << std::endl;
    }
    std::cout << "OAR filtering matrix, number of non-zero voxels: " << totalCount << std::endl;

    std::vector<int> h_filterOffsets(totalCount+1, 0);
    std::vector<int> h_filterColumns(totalCount, 0);
    std::vector<float> h_filterValues(totalCount, 1.0f);
    int idx = 0;
    for (int i=0; i<structs.size(); i++) {
        const StructInfo& currentStruct = structs[i];
        if (currentStruct.maxWeights < eps_fastdose &&
            currentStruct.minDoseTargetWeights < eps_fastdose &&
            currentStruct.OARWeights < eps_fastdose) {
            continue;
        }
        const std::vector<uint8_t>& mask = currentStruct.mask;
        for (size_t j=0; j<nVoxels; j++) {
            if (mask[j] > 0) {
                h_filterColumns[idx] = j;
                idx++;
            }
        }
    }
    for (int i=0; i<totalCount+1; i++)
        h_filterOffsets[i] = i;
    checkCudaErrors(hipMalloc((void**)&matFilterT.d_csr_offsets, (totalCount+1)*sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&matFilterT.d_csr_columns, totalCount*sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&matFilterT.d_csr_values, totalCount*sizeof(float)));
    checkCudaErrors(hipMemcpy(matFilterT.d_csr_offsets, h_filterOffsets.data(),
        h_filterOffsets.size()*sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(matFilterT.d_csr_columns, h_filterColumns.data(),
        h_filterColumns.size()*sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(matFilterT.d_csr_values, h_filterValues.data(),
        h_filterValues.size()*sizeof(float), hipMemcpyHostToDevice));
    matFilterT.numRows = totalCount;
    matFilterT.numCols = nVoxels;
    matFilterT.nnz = totalCount;
    checkCusparse(hipsparseCreateCsr(&matFilterT.matA, totalCount, nVoxels, totalCount,
        matFilterT.d_csr_offsets, matFilterT.d_csr_columns, matFilterT.d_csr_values,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));


    // initialize matFilter
    checkCudaErrors(hipMalloc((void**)&matFilter.d_csr_offsets, (nVoxels+1)*sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&matFilter.d_csr_columns, totalCount*sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&matFilter.d_csr_values, totalCount*sizeof(float)));
    matFilter.numRows = nVoxels;
    matFilter.numCols = totalCount;
    matFilter.nnz = totalCount;
    checkCusparse(hipsparseCreateCsr(&matFilter.matA, nVoxels, totalCount, totalCount,
        matFilter.d_csr_offsets, matFilter.d_csr_columns, matFilter.d_csr_values,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    size_t transposeBufferSize;
    void* transposeBuffer = nullptr;
    hipsparseHandle_t handle = nullptr;
    checkCusparse(hipsparseCreate(&handle));
    checkCusparse(hipsparseCsr2cscEx2_bufferSize(
        handle, matFilterT.numRows, matFilterT.numCols, matFilterT.nnz,
        matFilterT.d_csr_values, matFilterT.d_csr_offsets, matFilterT.d_csr_columns,
        matFilter.d_csr_values, matFilter.d_csr_offsets, matFilter.d_csr_columns,
        HIP_R_32F, HIPSPARSE_ACTION_SYMBOLIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG_DEFAULT,
        &transposeBufferSize));
    checkCudaErrors(hipMalloc((void**)&transposeBuffer, transposeBufferSize));
    checkCusparse(hipsparseCsr2cscEx2(
        handle, matFilterT.numRows, matFilterT.numCols, matFilter.nnz,
        matFilterT.d_csr_values, matFilterT.d_csr_offsets, matFilterT.d_csr_columns,
        matFilter.d_csr_values, matFilter.d_csr_offsets, matFilter.d_csr_columns,
        HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG_DEFAULT,
        transposeBuffer));
    checkCudaErrors(hipFree(transposeBuffer));
    checkCusparse(hipsparseDestroy(handle));
    return 0;
}


bool IMRT::OARFiltering(std::vector<MatCSR32>& OARMatricesT,
    const std::vector<MatCSR32>& matricesT,
    const MatCSR32& matFilter, const MatCSR32& matFilterT,
    int** d_bufferOffsets, int** d_bufferColumns, float** d_bufferValues
) {
    int numMatrices = matricesT.size();
    if (numMatrices != OARMatricesT.size()) {
        std::cerr << "The size of the two vectors, matricesT and "
            "OARMatricesT, are not the same." << std::endl;
        return 1;
    }
    hipsparseHandle_t handle;
    hipsparseSpGEMMDescr_t SpGEMMDesc;
    checkCusparse(hipsparseCreate(&handle));
    checkCusparse(hipsparseSpGEMM_createDescr(&SpGEMMDesc));

    // first, we know the number of rows of the result.
    // So we can pre allocate d_bufferOffsets
    std::vector<int> offsetIdx(numMatrices);
    int offsetIdxPrev = 0;
    for (int i=0; i<numMatrices; i++) {
        offsetIdx[i] = offsetIdxPrev;
        offsetIdxPrev += matricesT[i].numRows + 1;
    }
    checkCudaErrors(hipMalloc((void**)d_bufferOffsets, offsetIdxPrev*sizeof(int)));

    // pre-allocate buffer
    size_t nnz_prev = 0;
    for (int i=0; i<numMatrices; i++)
        nnz_prev += matricesT[i].nnz;
    int nnz_current = static_cast<int>(nnz_prev * 0.5f);
    std::cout << "The number of non-zero elements in the full matrix is: "
        << nnz_prev << ", we pre-allocate a buffer of size: " << nnz_current
        << " elements." << std::endl;
    checkCudaErrors(hipMalloc((void**)d_bufferColumns, nnz_current*sizeof(int)));
    checkCudaErrors(hipMalloc((void**)d_bufferValues, nnz_current*sizeof(float)));

    for (int i=0; i<numMatrices; i++) {
        MatCSR32& destMat = OARMatricesT[i];
        destMat.d_csr_offsets = *d_bufferOffsets + offsetIdx[i];
        int destNumRows = matricesT[i].numRows;
        checkCusparse(hipsparseCreateCsr(&destMat.matA, destNumRows, matFilter.numCols, 0,
            destMat.d_csr_offsets, nullptr, nullptr,
            HIPSPARSE_INDEX_32I,  HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    }

    // ask bufferSize1 bytes for external memory
    hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    float alpha = 1.0f;
    float beta = 0.0f;
    size_t bufferSize1 = 0;
    size_t bufferSize2 = 0;
    void* buffer1 = nullptr;
    void* buffer2 = nullptr;

    int cumuNnz = 0;
    for (int i=0; i<numMatrices; i++) {
        MatCSR32& dstMat = OARMatricesT[i];
        const MatCSR32& srcMat = matricesT[i];

        size_t bufferSize1Local = 0;
        checkCusparse(hipsparseSpGEMM_workEstimation(
            handle, opA, opB,
            &alpha, srcMat.matA, matFilter.matA, &beta, dstMat.matA,
            HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT,
            SpGEMMDesc, &bufferSize1Local, nullptr));
        
        if (bufferSize1Local > bufferSize1) {
            bufferSize1 = 2 * bufferSize1Local;
            if (buffer1 != nullptr)
                checkCudaErrors(hipFree(buffer1));
            checkCudaErrors(hipMalloc(&buffer1, bufferSize1));
            std::cout << "Buffer 1 resized: " << bufferSize1 << std::endl;
        }

        checkCusparse(hipsparseSpGEMM_workEstimation(
            handle, opA, opB,
            &alpha, srcMat.matA, matFilter.matA, &beta, dstMat.matA,
            HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT,
            SpGEMMDesc, &bufferSize1Local, buffer1));

        size_t bufferSize2Local = 0;
        checkCusparse(hipsparseSpGEMM_compute(
            handle, opA, opB,
            &alpha, srcMat.matA, matFilter.matA, &beta, dstMat.matA,
            HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT,
            SpGEMMDesc, &bufferSize2Local, nullptr));

        if (bufferSize2Local > bufferSize2) {
            bufferSize2 = 2 * bufferSize2Local;
            if (buffer2 != nullptr)
                checkCudaErrors(hipFree(buffer2));
            checkCudaErrors(hipMalloc(&buffer2, bufferSize2));
            std::cout << "Buffer 2 resized: " << bufferSize2 << std::endl;
        }

        checkCusparse(hipsparseSpGEMM_compute(
            handle, opA, opB,
            &alpha, srcMat.matA, matFilter.matA, &beta, dstMat.matA,
            HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT,
            SpGEMMDesc, &bufferSize2Local, buffer2
        ));

        int64_t C_num_rows1, C_num_cols1;
        checkCusparse(hipsparseSpMatGetSize(dstMat.matA,
            &C_num_rows1, &C_num_cols1, &dstMat.nnz));
        checkCusparse(hipsparseCsrSetPointers(dstMat.matA, dstMat.d_csr_offsets,
            *d_bufferColumns + cumuNnz, *d_bufferValues + cumuNnz));
        cumuNnz += dstMat.nnz;

        if (cumuNnz > nnz_current) {
            std::cerr << "The number of non-zero elements " << cumuNnz
                << " has exceeded the pre-allocated buffer " << nnz_current << std::endl;
            return 1;
        }

        checkCusparse(hipsparseSpGEMM_copy(handle, opA, opB,
            &alpha, srcMat.matA, matFilter.matA, &beta, dstMat.matA,
            HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT, SpGEMMDesc));

        std::cout << "Matrix " << i << " / " << numMatrices << " finished." << std::endl;
    }
    checkCudaErrors(hipFree(buffer2));
    checkCudaErrors(hipFree(buffer1));

    return 0;
}