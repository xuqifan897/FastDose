#include "hip/hip_runtime.h"
#include "IMRTDoseMat.cuh"
#include "IMRTDoseMatEigen.cuh"
#include "IMRTOptimize.cuh"
#include "IMRTOptimize_var.cuh"
#include "IMRTOptimize_var.h"
#include <iomanip>
#include <chrono>
#include <boost/filesystem.hpp>
namespace fs = boost::filesystem;

bool IMRT::OARFiltering(
    const std::string& resultFolder, const std::vector<StructInfo>& structs,
    std::vector<MatCSR_Eigen>& VOIMatrices, std::vector<MatCSR_Eigen>& VOIMatricesT,
    Weights_h& weights
) {
    MatCSR_Eigen filter, filterT;
    if (getStructFilter(filter, filterT, structs, weights)) {
        std::cerr << "OAR filter and its transpose construction error." << std::endl;
        return 1;
    }
    if (parallelSpGEMM(resultFolder, filter, filterT, VOIMatrices, VOIMatricesT)) {
        std::cerr << "CPU VOI dose loading matrices and their transpose "
            "construction error." << std::endl;
        return 1;
    }
    return 0;
}


size_t IMRT::sizeEstimate(
    const std::vector<MatCSR_Eigen>& VOIMatrices,
    const std::vector<MatCSR_Eigen>& VOIMatricesT,
    const std::vector<MatCSR_Eigen>& SpFluenceGrad,
    const std::vector<MatCSR_Eigen>& SpFluenceGradT) {
    
    std::vector<const std::vector<MatCSR_Eigen>*> iterator
        {&VOIMatrices, &VOIMatricesT, &SpFluenceGrad, &SpFluenceGradT};
    
    size_t size = 0;
    for (const std::vector<MatCSR_Eigen>* ptr : iterator) {
        int numMatrices = ptr->size();
        for (int i=0; i<numMatrices; i++) {
            const MatCSR_Eigen& localMat = (*ptr)[i];
            size_t localRows = localMat.getRows();
            size_t localNnz = localMat.getNnz();
            size_t localSize = (localRows + 1 + localNnz) * sizeof(size_t)
                + localNnz * sizeof(float);
            size += localSize;
        }
    }
    return size;
}


bool IMRT::MatReservior::load(const std::vector<MatCSR_Eigen>& source) {
    this->reservior.resize(source.size());
    for (int i=0; i<source.size(); i++) {
        MatCSR64& current = this->reservior[i];
        current.numRows = 0;
        current.numCols = 0;
        current.nnz = 0;
        current.matA = nullptr;
        current.d_csr_offsets = nullptr;
        current.d_csr_columns = nullptr;
        current.d_csr_values = nullptr;

        if(Eigen2Cusparse(source[i], current))
            return 1;
    }
    return 0;
}


bool IMRT::MatReservior::assemble_row_block(MatCSR64& target,
    const std::vector<uint8_t>& flags ) const {
    // firstly, check if the target is empty
    if (target.matA != nullptr || target.d_csr_offsets != nullptr ||
        target.d_csr_columns != nullptr || target.d_csr_values != nullptr ||
        target.d_buffer_spmv != nullptr) {
        std::cerr << "The target is not an empty matrix." << std::endl;
        return 1;
    }
    if (flags.size() != this->reservior.size()) {
        std::cerr << "The size of the input vector flags should be the same "
            "as this->reservior." << std::endl;
        return 1;
    }

    size_t total_nnz = 0;
    size_t total_rows = 0;
    size_t columns = 0;
    for (int i=0; i<this->reservior.size(); i++) {
        if (flags[i] == 0)
            continue;
        const MatCSR64& res = this->reservior[i];
        total_nnz += res.nnz;
        total_rows += res.numRows;
        if (columns == 0)
            columns = res.numCols;
        else if (columns != res.numCols) {
            std::cerr << "The number of columns inconsistent amoung matrices." << std::endl;
            return 1;
        }
    }

    target.numRows = total_rows;
    target.numCols = columns;
    target.nnz = total_nnz;
    checkCudaErrors(hipMalloc((void**)&target.d_csr_offsets, (total_rows+1)*sizeof(size_t)));
    checkCudaErrors(hipMalloc((void**)&target.d_csr_columns, total_nnz*sizeof(size_t)));
    checkCudaErrors(hipMalloc((void**)&target.d_csr_values, total_nnz*sizeof(float)));

    checkCudaErrors(hipMemset(target.d_csr_offsets, 0, sizeof(size_t)));
    size_t nnz_offset = 0;
    size_t row_offset = 0;
    std::vector<size_t> cumu_row;
    std::vector<size_t> cumu_nnz;
    cumu_row.reserve(this->reservior.size()+1);
    cumu_nnz.reserve(this->reservior.size()+1);
    cumu_row.push_back(0);
    cumu_nnz.push_back(0);
    for (int i=0; i<this->reservior.size(); i++) {
        if (flags[i] == 0)
            continue;
        const MatCSR64& res = this->reservior[i];
        size_t local_nnz = res.nnz;
        size_t local_rows = res.numRows;
        const size_t* local_offsets = res.d_csr_offsets;
        const size_t* local_columns = res.d_csr_columns;
        const float* local_values = res.d_csr_values;

        checkCudaErrors(hipMemcpyAsync(target.d_csr_columns + nnz_offset, local_columns,
            local_nnz*sizeof(size_t), hipMemcpyDeviceToDevice));
        checkCudaErrors(hipMemcpyAsync(target.d_csr_values + nnz_offset, local_values,
            local_nnz*sizeof(float), hipMemcpyDeviceToDevice));
        checkCudaErrors(hipMemcpyAsync(target.d_csr_offsets + row_offset + 1, local_offsets + 1,
            local_rows*sizeof(size_t), hipMemcpyDeviceToDevice));
        
        nnz_offset += local_nnz;
        row_offset += local_rows;

        cumu_row.push_back(row_offset);
        cumu_nnz.push_back(nnz_offset);
    }

    size_t* cumu_row_d = nullptr;
    size_t* cumu_nnz_d = nullptr;
    checkCudaErrors(hipMalloc((void**)&cumu_row_d, cumu_row.size()*sizeof(size_t)));
    checkCudaErrors(hipMalloc((void**)&cumu_nnz_d, cumu_nnz.size()*sizeof(size_t)));
    checkCudaErrors(hipMemcpy(cumu_row_d, cumu_row.data(),
        cumu_row.size()*sizeof(size_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(cumu_nnz_d, cumu_nnz.data(),
        cumu_nnz.size()*sizeof(size_t), hipMemcpyHostToDevice));
    
    dim3 blockSize(64, 1, 1);
    dim3 gridSize(1, 1, 1);
    gridSize.x = (cumu_row.size() - 1 + blockSize.x - 1) / blockSize.x;
    d_assembly_row_block<<<gridSize, blockSize>>>(
        target.d_csr_offsets, cumu_row_d, cumu_nnz_d, cumu_row.size()-1);
    checkCudaErrors(hipDeviceSynchronize());

    checkCusparse(hipsparseCreateCsr(
        &target.matA, target.numRows, target.numCols, target.nnz,
        target.d_csr_offsets, target.d_csr_columns, target.d_csr_values,
        HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    // clean up
    checkCudaErrors(hipFree(cumu_row_d));
    checkCudaErrors(hipFree(cumu_nnz_d));
    return 0;
}


__global__ void
IMRT::d_assembly_row_block(size_t* d_csr_offsets,
    size_t* cumu_row, size_t* cumu_nnz, size_t numMatrices) {
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= numMatrices)
        return;

    size_t row_start = cumu_row[idx];
    size_t row_end = cumu_row[idx + 1];
    size_t nnz_offset = cumu_nnz[idx];

    for (size_t i=row_start; i<row_end; i++) {
        size_t local = d_csr_offsets[i+1] + nnz_offset;
        d_csr_offsets[i+1] = local;
    }
}


bool IMRT::MatReservior_dev(
    const std::vector<MatCSR_Eigen>& VOIMatrices,
    const std::vector<MatCSR_Eigen>& VOIMatricesT,
    const std::vector<MatCSR_Eigen>& SpFluenceGrad,
    const std::vector<MatCSR_Eigen>& SpFluenceGradT
) {
    IMRT::MatReservior VOIReservior, VOIReserviorT, FGReservior, FGReserviorT;
    #if true
    // estimate size
        size_t totalSize = IMRT::sizeEstimate(VOIMatrices, VOIMatricesT,
            SpFluenceGrad, SpFluenceGradT);
        std::cout << "Total size: " << (float)totalSize / (1<<30) << " G" << std::endl;
    #endif
    #if slicingTiming
        auto time0 = std::chrono::high_resolution_clock::now();
    #endif
    if (VOIReservior.load(VOIMatrices) ||
        VOIReserviorT.load(VOIMatricesT) ||
        FGReservior.load(SpFluenceGrad) ||
        FGReservior.load(SpFluenceGradT)) {
        std::cerr << "Loading data from CPU to GPU error." << std::endl;
    }
    #if slicingTiming
        auto time1 = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(time1 - time0);
        std::cout << std::setprecision(4) << "Loading data from CPU to GPU time elapsed: " 
            << duration.count() * 1e-6f << " [s]" << std::endl;
    #endif

    std::vector<uint8_t> flags(VOIReserviorT.reservior.size(), 1);
    MatCSR64 VOIMatT;
    if (VOIReserviorT.assemble_row_block(VOIMatT, flags))
        return 1;
    #if slicingTiming
        auto time2 = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::microseconds>(time2 - time1);
        std::cout << "Assembly_row_block time elapsed: "
            << duration.count() * 1e-6f << " [s]" << std::endl;
    #endif

    #if false
        // verify against MatCSR_Eigen
        std::cout << "\nATrans Benchmarking starts..." << std::endl;
        // for comparision
        MatCSR_Eigen VOIMat_Eigen;
        MatCSR_Eigen VOIMatT_Eigen;
        MatCSR_Eigen D_Eigen;
        MatCSR_Eigen DTrans_Eigen;
        std::vector<MatCSR_Eigen*> VOIMatrice_ptr(VOIMatricesT.size(), nullptr);
        std::vector<MatCSR_Eigen*> VOIMatriceT_ptr(VOIMatricesT.size(), nullptr);
        std::vector<MatCSR_Eigen*> SpFluenceGrad_ptr(VOIMatricesT.size(), nullptr);
        std::vector<MatCSR_Eigen*> SpFluenceGradT_ptr(VOIMatricesT.size(), nullptr);
        for (int i=0; i<VOIMatricesT.size(); i++) {
            VOIMatrice_ptr[i] = (MatCSR_Eigen*)&VOIMatrices[i];
            VOIMatriceT_ptr[i] = (MatCSR_Eigen*)&VOIMatricesT[i];
            SpFluenceGrad_ptr[i] = (MatCSR_Eigen*)&SpFluenceGrad[i];
            SpFluenceGradT_ptr[i] = (MatCSR_Eigen*)&SpFluenceGradT[i];
        }
        matFuseFunc(VOIMatrice_ptr, VOIMatriceT_ptr, SpFluenceGrad_ptr, SpFluenceGradT_ptr,
            VOIMat_Eigen, VOIMatT_Eigen, D_Eigen, DTrans_Eigen);

        std::vector<size_t> VOIMatT_offsets(VOIMatT.numRows + 1);
        std::vector<size_t> VOIMatT_columns(VOIMatT.nnz);
        std::vector<float> VOIMatT_values(VOIMatT.nnz);
        checkCudaErrors(hipMemcpy(VOIMatT_offsets.data(), VOIMatT.d_csr_offsets,
            VOIMatT_offsets.size()*sizeof(size_t), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(VOIMatT_columns.data(), VOIMatT.d_csr_columns,
            VOIMatT_columns.size()*sizeof(size_t), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(VOIMatT_values.data(), VOIMatT.d_csr_values,
            VOIMatT_values.size()*sizeof(float), hipMemcpyDeviceToHost));

        EigenIdxType* ref_offsets = *VOIMatT_Eigen.getOffset();
        const EigenIdxType* ref_columns = VOIMatT_Eigen.getIndices();
        const float* ref_values = VOIMatT_Eigen.getValues();

        for (size_t i=0; i<VOIMatT_offsets.size(); i++) {
            if (VOIMatT_offsets[i] != ref_offsets[i]) {
                std::cerr << "Offsets unmatch at i=" << i << ", VOIMatT_offsets[i]=="
                    << VOIMatT_offsets[i] << ", ref_offsets[i]==" << ref_offsets[i] << std::endl;
                return 1;
            }
        }
        for (size_t i=0; i<VOIMatT_columns.size(); i++) {
            if (VOIMatT_columns[i] != ref_columns[i] ||
                std::abs(VOIMatT_values[i] - ref_values[i]) > 1e-4f) {
                std::cerr << "Element unmatch at i=" << i << " test: (" << VOIMatT_columns[i]
                    << ", " << VOIMatT_values[i] << "), reference: (" << ref_columns[i]
                    << ", " << ref_values[i] << ")" << std::endl;
                return 1;
            }
        }
    #endif

    #if true
        // VOIMatT verification
        array_1d<float> input, output_full, output_slice;
        size_t sliceIdx = 63;
        const MatCSR64& slice = VOIReserviorT.reservior[sliceIdx];
        arrayInit(input, VOIMatT.numCols);
        arrayInit(output_full, VOIMatT.numRows);
        arrayInit(output_slice, slice.numRows);
        arrayRand01(input);

        hipsparseHandle_t handle;
        checkCusparse(hipsparseCreate(&handle));
        float alpha = 1.0f;
        float beta = 0.0f;

        void* buffer;
        size_t bufferSize;
        checkCusparse(hipsparseSpMV_bufferSize(
            handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, VOIMatT.matA, input.vec, &beta, output_full.vec,
            HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
        checkCudaErrors(hipMalloc((void**)&buffer, bufferSize));

        checkCusparse(hipsparseSpMV(
            handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, VOIMatT.matA, input.vec, &beta, output_full.vec,
            HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer));
        checkCusparse(hipsparseSpMV(
            handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, slice.matA, input.vec, &beta, output_slice.vec,
            HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer));

        // compare results
        size_t resultOffset = 0;
        for (int i=0; i<sliceIdx; i++)
            resultOffset += VOIReserviorT.reservior[i].numRows;
        std::vector<float> result_full(output_slice.size);
        std::vector<float> result_slice(output_slice.size);
        checkCudaErrors(hipMemcpy(result_full.data(), output_full.data + resultOffset,
            result_full.size()*sizeof(float), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(result_slice.data(), output_slice.data,
            result_slice.size()*sizeof(float), hipMemcpyDeviceToHost));
        
        for (size_t i=0; i<result_full.size(); i++) {
            if (abs(result_full[i] - result_slice[i]) > 1e-4f * result_slice[i]) {
                std::cerr << "Result unmatch at index: " << i << ", result_full[i] == "
                    << result_full[i] << ", result_slice[i] == " << result_slice[i] << std::endl;
                return 1;
            }
        }
        checkCudaErrors(hipFree(buffer));
        checkCusparse(hipsparseDestroy(handle));

        std::cout << "assemble_row_block passed the test!" << std::endl;
    #endif

    return 0;
}


bool IMRT::MatReservior::assemble_col_block(MatCSR64& target,
    const std::vector<MatCSR_Eigen>& reservior_h,
    const std::vector<uint8_t>& flags) const {
    // firstly, check if the target is empty
    if (target.matA != nullptr || target.d_csr_offsets != nullptr ||
        target.d_csr_columns != nullptr || target.d_csr_values != nullptr ||
        target.d_buffer_spmv != nullptr) {
        std::cerr << "The target is not an empty matrix." << std::endl;
        return 1;
    }
    if (flags.size() != this->reservior.size()) {
        std::cerr << "The size of the input vector flags should be the same "
            "as this->reservior." << std::endl;
        return 1;
    }
    size_t numRows = 0;
    size_t numCols = 0;
    size_t total_nnz = 0;
    size_t numMatrices = 0;
    std::vector<size_t> cumu_nnz;
    #if false
        auto time0 = std::chrono::high_resolution_clock::now();
    #endif
    assemble_col_block_meta(numRows, numCols, total_nnz, numMatrices,
        cumu_nnz, flags, reservior_h);
    #if false
        auto time1 = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(time1 - time0);
        std::cout << "Function assemble_col_block_meta time elapsed: "
            << duration.count() * 1e-6f << " [s]" << std::endl;
    #endif

    target.numRows = numRows;
    target.numCols = numCols;
    target.nnz = total_nnz;
    checkCudaErrors(hipMalloc((void**)&target.d_csr_offsets, (target.numRows+1)*sizeof(size_t)));
    checkCudaErrors(hipMalloc((void**)&target.d_csr_columns, target.nnz*sizeof(size_t)));
    checkCudaErrors(hipMalloc((void**)&target.d_csr_values, target.nnz*sizeof(float)));

    if (this->reservior.size() != reservior_h.size()) {
        std::cerr << "this->reservior.size() != reservior_h.size()" << std::endl;
        return 1;
    }

    checkCudaErrors(hipMemcpy(target.d_csr_offsets, cumu_nnz.data(),
        (target.numRows+1)*sizeof(size_t), hipMemcpyHostToDevice));
    
    size_t** source_offsets = nullptr;
    size_t** source_columns = nullptr;
    float** source_values = nullptr;
    size_t* source_columns_offset = nullptr;
    checkCudaErrors(hipMalloc((void**)&source_offsets, numMatrices*sizeof(size_t*)));
    checkCudaErrors(hipMalloc((void**)&source_columns, numMatrices*sizeof(size_t*)));
    checkCudaErrors(hipMalloc((void**)&source_values, numMatrices*sizeof(float*)));
    checkCudaErrors(hipMalloc((void**)&source_columns_offset, numMatrices*sizeof(size_t)));
    std::vector<size_t*> h_source_offsets(numMatrices, nullptr);
    std::vector<size_t*> h_source_columns(numMatrices, nullptr);
    std::vector<float*> h_source_values(numMatrices, nullptr);
    std::vector<size_t> h_source_columns_offset(numMatrices + 1, 0);
    int idx = 0;
    for (int i=0; i<this->reservior.size(); i++) {
        if (flags[i] == 0)
            continue;
        const MatCSR64& res = this->reservior[i];
        h_source_offsets[idx] = res.d_csr_offsets;
        h_source_columns[idx] = res.d_csr_columns;
        h_source_values[idx] = res.d_csr_values;
        h_source_columns_offset[idx + 1] = h_source_columns_offset[idx] + res.numCols;
        idx ++;
    }
    checkCudaErrors(hipMemcpy(source_offsets, h_source_offsets.data(),
        numMatrices*sizeof(size_t*), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(source_columns, h_source_columns.data(),
        numMatrices*sizeof(size_t*), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(source_values, h_source_values.data(),
        numMatrices*sizeof(float*), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(source_columns_offset, h_source_columns_offset.data(),
        numMatrices*sizeof(size_t), hipMemcpyHostToDevice));

    dim3 blockSize(64, 1, 1);
    dim3 gridSize(1, 1, 1);
    gridSize.x = (target.numRows + blockSize.x - 1) / blockSize.x;
    d_assembly_col_block<<<gridSize, blockSize>>>(
        target.d_csr_offsets, target.d_csr_columns, target.d_csr_values,
        source_offsets, source_columns, source_values,
        source_columns_offset, numRows, numMatrices);
    checkCudaErrors(hipDeviceSynchronize());

    checkCusparse(hipsparseCreateCsr(
        &target.matA, target.numRows, target.numCols, target.nnz,
        target.d_csr_offsets, target.d_csr_columns, target.d_csr_values,
        HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    // clean up
    checkCudaErrors(hipFree(source_offsets));
    checkCudaErrors(hipFree(source_columns));
    checkCudaErrors(hipFree(source_values));
    checkCudaErrors(hipFree(source_columns_offset));
    return 0;
}


__global__ void
IMRT::d_assembly_col_block(size_t* d_csr_offsets, size_t* d_csr_columns, float* d_csr_values,
    size_t** source_offsets, size_t** source_columns, float** source_values,
    size_t* source_columns_offset, size_t numRows, size_t numMatrices) {
    size_t row_idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (row_idx >= numRows)
        return;
    
    size_t idx_start = d_csr_offsets[row_idx];
    for (size_t i=0; i<numMatrices; i++) {
        size_t* current_source_offsets = source_offsets[i];
        size_t* current_source_columns = source_columns[i];
        float* current_source_values = source_values[i];
        size_t current_columns_offset = source_columns_offset[i];

        size_t current_idx_start = current_source_offsets[row_idx];
        size_t current_idx_end = current_source_offsets[row_idx+1];
        for (size_t element_idx=current_idx_start; element_idx<current_idx_end; element_idx++) {
            d_csr_columns[idx_start] = current_source_columns[element_idx] + current_columns_offset;
            d_csr_values[idx_start] = current_source_values[element_idx];
            idx_start ++;
        }
    }
}


bool IMRT::MatReservior_dev_col(
    const std::vector<MatCSR_Eigen>& VOIMatrices,
    const std::vector<MatCSR_Eigen>& VOIMatricesT,
    const std::vector<MatCSR_Eigen>& SpFluenceGrad,
    const std::vector<MatCSR_Eigen>& SpFluenceGradT
) {
    IMRT::MatReservior VOIReservior, VOIReserviorT, FGReservior, FGReserviorT;
    #if true
    // estimate size
        size_t totalSize = IMRT::sizeEstimate(VOIMatrices, VOIMatricesT,
            SpFluenceGrad, SpFluenceGradT);
        std::cout << "Total size: " << (float)totalSize / (1<<30) << " G" << std::endl;
    #endif
    #if slicingTiming
        auto time0 = std::chrono::high_resolution_clock::now();
    #endif
    if (VOIReservior.load(VOIMatrices) ||
        VOIReserviorT.load(VOIMatricesT) ||
        FGReservior.load(SpFluenceGrad) ||
        FGReservior.load(SpFluenceGradT)) {
        std::cerr << "Loading data from CPU to GPU error." << std::endl;
    }
    #if slicingTiming
        auto time1 = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(time1 - time0);
        std::cout << std::setprecision(4) << "Loading data from CPU to GPU time elapsed: " 
            << duration.count() * 1e-6f << " [s]" << std::endl;
    #endif
    std::vector<uint8_t> flags(VOIReservior.reservior.size(), 1);
    MatCSR64 VOIMat;
    if (VOIReservior.assemble_col_block(VOIMat, VOIMatrices, flags))
        return 1;
    #if slicingTiming
        auto time2 = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::microseconds>(time2 - time1);
        std::cout << "Assembly_col_block time elapsed: "
            << duration.count() * 1e-6f << " [s]" << std::endl;
    #endif

    #if false
        // verify against MatCSR_Eigen
        std::cout << "\nATrans Benchmarking starts..." << std::endl;
        // for comparision
        MatCSR_Eigen VOIMat_Eigen;
        MatCSR_Eigen VOIMatT_Eigen;
        MatCSR_Eigen D_Eigen;
        MatCSR_Eigen DTrans_Eigen;
        std::vector<MatCSR_Eigen*> VOIMatrice_ptr(VOIMatricesT.size(), nullptr);
        std::vector<MatCSR_Eigen*> VOIMatriceT_ptr(VOIMatricesT.size(), nullptr);
        std::vector<MatCSR_Eigen*> SpFluenceGrad_ptr(VOIMatricesT.size(), nullptr);
        std::vector<MatCSR_Eigen*> SpFluenceGradT_ptr(VOIMatricesT.size(), nullptr);
        for (int i=0; i<VOIMatricesT.size(); i++) {
            VOIMatrice_ptr[i] = (MatCSR_Eigen*)&VOIMatrices[i];
            VOIMatriceT_ptr[i] = (MatCSR_Eigen*)&VOIMatricesT[i];
            SpFluenceGrad_ptr[i] = (MatCSR_Eigen*)&SpFluenceGrad[i];
            SpFluenceGradT_ptr[i] = (MatCSR_Eigen*)&SpFluenceGradT[i];
        }
        matFuseFunc(VOIMatrice_ptr, VOIMatriceT_ptr, SpFluenceGrad_ptr, SpFluenceGradT_ptr,
            VOIMat_Eigen, VOIMatT_Eigen, D_Eigen, DTrans_Eigen);

        std::vector<size_t> VOIMat_offsets(VOIMat.numRows + 1);
        std::vector<size_t> VOIMat_columns(VOIMat.nnz);
        std::vector<float> VOIMat_values(VOIMat.nnz);
        checkCudaErrors(hipMemcpy(VOIMat_offsets.data(), VOIMat.d_csr_offsets,
            VOIMat_offsets.size()*sizeof(size_t), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(VOIMat_columns.data(), VOIMat.d_csr_columns,
            VOIMat_columns.size()*sizeof(size_t), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(VOIMat_values.data(), VOIMat.d_csr_values,
            VOIMat_values.size()*sizeof(float), hipMemcpyDeviceToHost));

        EigenIdxType* ref_offsets = *VOIMat_Eigen.getOffset();
        const EigenIdxType* ref_columns = VOIMat_Eigen.getIndices();
        const float* ref_values = VOIMat_Eigen.getValues();

        for(size_t i=0; i<VOIMat_offsets.size(); i++) {
            if (VOIMat_offsets[i] != ref_offsets[i]) {
                std::cerr << "Offsets unmatch at i=" << i << ", VOIMat_offsets[i]=="
                    << VOIMat_offsets[i] << ", ref_offsets[i]==" << ref_offsets[i] << std::endl;
                return 1;
            }
        }

        for (size_t i=0; i<VOIMat_columns.size(); i++) {
            if (VOIMat_columns[i] != ref_columns[i] ||
                std::abs(VOIMat_values[i] - ref_values[i]) > 1e-4f) {
                std::cerr << "Element unmatch at i=" << i << ", test: (" << VOIMat_columns[i]
                    << ", " << VOIMat_values[i] << "), reference: (" << ref_columns[i]
                    << ", " << ref_values[i] << ")" << std::endl;
                return 1;
            }
        }
    std::cout << "assemble_col_block passed the test!" << std::endl;
    #endif

    return 0;
}