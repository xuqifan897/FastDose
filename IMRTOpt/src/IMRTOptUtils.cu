#include <iostream>
#include "IMRTOptimize.cuh"

template <class T>
IMRT::array_1d<T>& IMRT::array_1d<T>::operator=(const IMRT::array_1d<T>& other) {
    if (this != &other) {  // Avoid self-assignment
        if (this->size == other.size) {
            // of the same size, no need to allocate memory
            checkCudaErrors(hipMemcpy(this->data, other.data,
                other.size*sizeof(float), hipMemcpyDeviceToDevice));
            if (other.vec != nullptr && this->vec == nullptr)
                checkCusparse(hipsparseCreateDnVec(&this->vec, this->size, this->data, HIP_R_32F));
        } else {
            if (this->vec != nullptr)
                checkCusparse(hipsparseDestroyDnVec(this->vec));
            if (this->data != nullptr)
                checkCudaErrors(hipFree(this->data));
            
            this->size = other.size;
            checkCudaErrors(hipMalloc((void**)&this->data, this->size*sizeof(float)));
            checkCudaErrors(hipMemcpy(this->data, other.data,
                this->size*sizeof(float), hipMemcpyDeviceToDevice));
            if (other.vec != nullptr)
                checkCusparse(hipsparseCreateDnVec(&this->vec, this->size, this->data, HIP_R_32F));
        }
    }
    return *this;
}


IMRT::eval_g::eval_g(size_t ptv_voxels, size_t oar_voxels) {
    this->PTV_voxels = ptv_voxels;
    this->OAR_voxels = oar_voxels;

    checkCudaErrors(hipMalloc((void**)this->Ax.data, (ptv_voxels+oar_voxels)*sizeof(float)));
    this->Ax.size = ptv_voxels + oar_voxels;
    checkCusparse(hipsparseCreateDnVec(&this->Ax.vec, this->Ax.size, this->Ax.data, HIP_R_32F));

    // checkCudaErrors()
}


bool IMRT::assignmentTest(){
    array_1d<float> source;
    source.size = 4;
    std::vector<float> source_data_h {1.0f, 2.0f, 3.0f, 4.0f};
    checkCudaErrors(hipMalloc((void**)&source.data, source.size*sizeof(float)));
    checkCudaErrors(hipMemcpy(source.data, source_data_h.data(),
        source.size*sizeof(float), hipMemcpyHostToDevice));

    array_1d<float> dest;
    dest = source;
    std::vector<float> dest_data_h(dest.size);
    checkCudaErrors(hipMemcpy(dest_data_h.data(), dest.data,
        dest.size * sizeof(float), hipMemcpyDeviceToHost));
    
    std::cout << "Array dest size: " << dest.size << std::endl;
    for (int i=0; i<dest.size; i++)
        std::cout << dest_data_h[i] << " ";
    std::cout << std::endl;
    return 0;
}