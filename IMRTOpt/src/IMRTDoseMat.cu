#include "hip/hip_runtime.h"
#include <fstream>
#include <string>
#include <iomanip>
#include <limits>
#include <boost/filesystem.hpp>
namespace fs = boost::filesystem;

#include "IMRTDoseMat.cuh"
#include "IMRTArgs.h"
#include "IMRTgeom.cuh"

namespace fd = fastdose;

bool IMRT::DoseMatConstruction(
    std::vector<BeamBundle>& beam_bundles,
    fd::DENSITY_d& density_d,
    fd::SPECTRUM_h& spectrum_h,
    fd::KERNEL_h& kernel_h,
    MatCSREnsemble** matEns,
    hipStream_t stream
) {
    hipEvent_t start, stop, globalStart, globalStop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventCreate(&globalStart));
    checkCudaErrors(hipEventCreate(&globalStop));
    float milliseconds;

    // To calculate the beamlet dose of several beams together.
    // When concurrency is set to 1, calculate the dose for one beam at a time.
    int concurrency = 1;
    float extent = getarg<float>("extent");
    int iterations = (beam_bundles.size() + concurrency - 1) / concurrency;
    std::vector<size_t> numRowsPerMat(iterations, 0);

    // get the maximum number of beamlets, and the maximum beamlet length
    int maxNumBeamletsPerBatch = 0;
    int maxBeamletLength = 0;
    for (int i=0; i<iterations; i++) {
        int localNumBeamlets = 0;
        int beam_bundle_idx_begin = i * concurrency;
        int beam_bundle_idx_end = (i + 1) * concurrency;
        beam_bundle_idx_end = min(beam_bundle_idx_end, (int)(beam_bundles.size()));
        for (int j=beam_bundle_idx_begin; j<beam_bundle_idx_end; j++) {
            localNumBeamlets += beam_bundles[j].beams_h.size();
            for (int k=0; k<beam_bundles[j].beams_h.size(); k++)
                maxBeamletLength = max(maxBeamletLength, beam_bundles[j].beams_h[k].long_dim);
        }
        maxNumBeamletsPerBatch = max(maxNumBeamletsPerBatch, localNumBeamlets);
        numRowsPerMat[i] = localNumBeamlets;
    }
    std::cout << std::endl << "Maximum number of beamlets per batch: " << maxNumBeamletsPerBatch
        << ", maximum beamlet length: " << maxBeamletLength << std::endl << std::endl;

    // prepare the sparse dose matrices
    uint3 densityDim = density_d.VolumeDim;
    size_t numDensityVoxels = densityDim.x * densityDim.y * densityDim.z;
    size_t EstNonZeroElementsPerMat = getarg<size_t>("EstNonZeroElementsPerMat");
    size_t estBufferSize = EstNonZeroElementsPerMat * beam_bundles.size();
    *matEns = new MatCSREnsemble(numRowsPerMat, numDensityVoxels, estBufferSize);

    // allocate working buffers
    // for safty check
    unsigned long long denseDoseMatSize_ = maxNumBeamletsPerBatch * densityDim.x * densityDim.y * densityDim.z;
    if (denseDoseMatSize_ > std::numeric_limits<uint>::max()) {
        std::cerr << "The size of the dense dose matrix is " << denseDoseMatSize_
            << ", which is beyond the range size_t can represent. "
            "Please reduce the concurrency parameter" << std::endl;
        return 1;
    }
    size_t denseDoseMatSize = maxNumBeamletsPerBatch * densityDim.x * densityDim.y * densityDim.z;
    float* d_denseDoseMat = nullptr;
    checkCudaErrors(hipMalloc((void**)&d_denseDoseMat, denseDoseMatSize*sizeof(float)));

    fd::d_BEAM_d* d_BeamletsBuffer = nullptr;
    float* d_FluenceBuffer = nullptr;
    float* d_DensityBEVBuffer = nullptr;
    float* d_TeramBEVBuffer = nullptr;
    float* d_DoseBEVBuffer = nullptr;
    int subFluenceDim = getarg<int>("subFluenceDim");
    int subFluenceOn = getarg<int>("subFluenceOn");
    size_t bufferSize = maxNumBeamletsPerBatch * subFluenceDim * subFluenceDim * maxBeamletLength;
    checkCudaErrors(hipMalloc((void**)(&d_BeamletsBuffer),
        maxNumBeamletsPerBatch * sizeof(fd::d_BEAM_d)));
    checkCudaErrors(hipMalloc((void**)(&d_FluenceBuffer), maxNumBeamletsPerBatch
        * subFluenceDim * subFluenceDim * sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&d_DensityBEVBuffer, bufferSize*sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&d_TeramBEVBuffer, bufferSize*sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&d_DoseBEVBuffer, bufferSize*sizeof(float)));

    // Fluence buffer can be initialized before-hands
    std::vector<float> h_FluenceBuffer(maxNumBeamletsPerBatch * subFluenceDim * subFluenceDim);
    std::vector<float> h_SingleFluence(subFluenceDim * subFluenceDim, 0.0f);
    int FmapLeadingX = static_cast<int>((subFluenceDim - subFluenceOn) * 0.5f);
    int FmapLeadingY = FmapLeadingX;
    for (int j=FmapLeadingY; j<FmapLeadingY + subFluenceOn; j++) {
        for (int i=FmapLeadingX; i<FmapLeadingX + subFluenceOn; i++) {
            int idx = i + j * subFluenceDim;
            h_SingleFluence[idx] = 1.0f;
        }
    }
    for (int j=0; j<maxNumBeamletsPerBatch; j++) {
        size_t globalOffset = j * subFluenceDim * subFluenceDim;
        for (int i=0; i<subFluenceDim*subFluenceDim; i++) {
            h_FluenceBuffer[i + globalOffset] = h_SingleFluence[i];
        }
    }
    checkCudaErrors(hipMemcpy(d_FluenceBuffer, h_FluenceBuffer.data(),
        maxNumBeamletsPerBatch*subFluenceDim*subFluenceDim*sizeof(float),
        hipMemcpyHostToDevice));


    // buffer array
    std::vector<float*> h_FluenceArray(maxNumBeamletsPerBatch, nullptr);
    for (int i=0; i<maxNumBeamletsPerBatch; i++)
        h_FluenceArray[i] = d_FluenceBuffer + i * subFluenceDim * subFluenceDim;
    std::vector<float*> h_DensityArray(maxNumBeamletsPerBatch, nullptr);
    std::vector<float*> h_TermaArray(maxNumBeamletsPerBatch, nullptr);
    std::vector<float*> h_DoseArray(maxNumBeamletsPerBatch, nullptr);
    for (int i=0; i<maxNumBeamletsPerBatch; i++) {
        size_t offset = i * subFluenceDim * subFluenceDim * maxBeamletLength;
        h_DensityArray[i] = d_DensityBEVBuffer + offset;
        h_TermaArray[i] = d_TeramBEVBuffer + offset;
        h_DoseArray[i] = d_DoseBEVBuffer + offset;
    }
    float** d_FluenceArray = nullptr;
    float** d_DensityArray = nullptr;
    float** d_TermaArray = nullptr;
    float** d_DoseArray = nullptr;
    checkCudaErrors(hipMalloc((void***)&d_FluenceArray, maxNumBeamletsPerBatch*sizeof(float*)));
    checkCudaErrors(hipMalloc((void***)&d_DensityArray, maxNumBeamletsPerBatch*sizeof(float*)));
    checkCudaErrors(hipMalloc((void***)&d_TermaArray, maxNumBeamletsPerBatch*sizeof(float*)));
    checkCudaErrors(hipMalloc((void***)&d_DoseArray, maxNumBeamletsPerBatch*sizeof(float*)));
    checkCudaErrors(hipMemcpy(d_FluenceArray, h_FluenceArray.data(),
        maxNumBeamletsPerBatch*sizeof(float*), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_DensityArray, h_DensityArray.data(),
        maxNumBeamletsPerBatch*sizeof(float*), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_TermaArray, h_TermaArray.data(),
        maxNumBeamletsPerBatch*sizeof(float*), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_DoseArray, h_DoseArray.data(),
        maxNumBeamletsPerBatch*sizeof(float*), hipMemcpyHostToDevice));

    // pack array
    int2 packDim;
    packDim.x = (int)ceilf(sqrtf((float)maxNumBeamletsPerBatch));
    packDim.y = (int)ceilf((float)maxNumBeamletsPerBatch / packDim.x);
    int3 packArrayDim {
        packDim.x * subFluenceDim,
        packDim.y * subFluenceDim,
        maxBeamletLength };
    size_t packArraySize = packArrayDim.x * packArrayDim.y * packArrayDim.z;
    float* packArray = nullptr;
    checkCudaErrors(hipMalloc((void**)&packArray, packArraySize*sizeof(float)));

    // for sampling
    dim3 samplingBlockSize{8, 8, 8};
    dim3 samplingGridSize {
        (densityDim.x - 1 + samplingBlockSize.x) / samplingBlockSize.x,
        (densityDim.y - 1 + samplingBlockSize.y) / samplingBlockSize.y,
        (densityDim.z - 1 + samplingBlockSize.z) / samplingBlockSize.z
    };
    size_t preSamplingGridSize = samplingGridSize.x * samplingGridSize.y
        * samplingGridSize.z * maxNumBeamletsPerBatch;
    bool* d_preSamplingArray = nullptr;
    checkCudaErrors(hipMalloc((void**)(&d_preSamplingArray), preSamplingGridSize*sizeof(bool)));

    // the array to store the long_dim of each beamlets
    int* d_beamletLongArray = nullptr;
    checkCudaErrors(hipMalloc((void**)&d_beamletLongArray, maxNumBeamletsPerBatch*sizeof(int)));

    // prepare texture components
    hipArray* DoseBEV_Arr;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipExtent volumeSize = make_hipExtent(packArrayDim.x, packArrayDim.y, packArrayDim.z);
    hipMalloc3DArray(&DoseBEV_Arr, &channelDesc, volumeSize);

    // create a stream for memory reset, so that it can overlap computation and memory operations
    hipStream_t memsetStream;
    checkCudaErrors(hipStreamCreate(&memsetStream));

    hipEventRecord(globalStart);
    // for debug purposes
    for (int i=0; i<iterations; i++) {
        hipEventRecord(start);

        // firstly, prepare beamlet information
        int localNumBeamlets = 0;
        int beam_bundle_idx_begin = i * concurrency;
        int beam_bundle_idx_end = (i + 1) * concurrency;
        beam_bundle_idx_end = min(beam_bundle_idx_end, (int)(beam_bundles.size()));
        for (int j=beam_bundle_idx_begin; j<beam_bundle_idx_end; j++)
            localNumBeamlets += beam_bundles[j].beams_h.size();
        
        std::vector<fd::d_BEAM_d> h_BeamletsBuffer;
        std::vector<int> h_beamletLongArray;
        h_BeamletsBuffer.reserve(localNumBeamlets);
        h_beamletLongArray.reserve(localNumBeamlets);
        size_t pitch = subFluenceDim * subFluenceDim * sizeof(float);
        for (int j=beam_bundle_idx_begin; j<beam_bundle_idx_end; j++) {
            for (int k=0; k<beam_bundles[j].beams_h.size(); k++) {
                const fd::BEAM_h& h_source = beam_bundles[j].beams_h[k];
                h_BeamletsBuffer.push_back(fd::d_BEAM_d(h_source, pitch, pitch));
                h_beamletLongArray.push_back(h_source.long_dim);
            }
        }
        checkCudaErrors(hipMemcpyAsync(d_BeamletsBuffer, h_BeamletsBuffer.data(),
            localNumBeamlets*sizeof(fd::d_BEAM_d), hipMemcpyHostToDevice, stream));
        checkCudaErrors(hipMemcpyAsync(d_beamletLongArray, h_beamletLongArray.data(),
            localNumBeamlets*sizeof(int), hipMemcpyHostToDevice, stream));
        
        // Calculate Terma
        size_t fmap_npixels = subFluenceDim * subFluenceDim;
        fd::TermaComputeCollective(
            fmap_npixels,
            localNumBeamlets,
            d_BeamletsBuffer,
            d_FluenceArray,
            d_TermaArray,
            d_DensityArray,
            density_d,
            spectrum_h,
            stream
        );
        
        // Calculate Dose
        fd::DoseComputeCollective(
            fmap_npixels,
            localNumBeamlets,
            d_BeamletsBuffer,
            d_TermaArray,
            d_DensityArray,
            d_DoseArray,
            kernel_h.nTheta,
            kernel_h.nPhi,
            stream);
        
        // Copy the result to packed array
        BEV2PVCSInterp(
            d_denseDoseMat,
            denseDoseMatSize,
            d_BeamletsBuffer,
            localNumBeamlets,
            density_d,
            d_DoseArray,
            pitch / sizeof(float),
            d_preSamplingArray,
            preSamplingGridSize,
            packArray,
            packDim,
            make_int2(subFluenceDim, subFluenceDim),
            packArrayDim,
            &DoseBEV_Arr,
            d_beamletLongArray,
            extent,
            stream,
            memsetStream
        );

        (**matEns).addMat(d_denseDoseMat, localNumBeamlets, numDensityVoxels);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        std::cout << "Iteration: " << i << ", beam bundle indices: "
            << beam_bundle_idx_begin << " ~ " << beam_bundle_idx_end - 1
            << " / " << beam_bundles.size() << ", time elapsed: "
            << milliseconds << " [ms]" << std::endl;
    }
    checkCudaErrors(hipStreamDestroy(memsetStream));

    hipEventRecord(globalStop);
    hipEventSynchronize(globalStop);
    hipEventElapsedTime(&milliseconds, globalStart, globalStop);
    std::cout << "Dose calculation time: " << milliseconds * 0.001f << " s" << std::endl;

    // clean up
    checkCudaErrors(hipFreeArray(DoseBEV_Arr));
    checkCudaErrors(hipFree(d_beamletLongArray));
    checkCudaErrors(hipFree(d_preSamplingArray));
    checkCudaErrors(hipFree(packArray));

    checkCudaErrors(hipFree(d_DoseArray));
    checkCudaErrors(hipFree(d_TermaArray));
    checkCudaErrors(hipFree(d_DensityArray));
    checkCudaErrors(hipFree(d_FluenceArray));

    checkCudaErrors(hipFree(d_DoseBEVBuffer));
    checkCudaErrors(hipFree(d_TeramBEVBuffer));
    checkCudaErrors(hipFree(d_DensityBEVBuffer));
    checkCudaErrors(hipFree(d_FluenceBuffer));
    checkCudaErrors(hipFree(d_BeamletsBuffer));

    checkCudaErrors(hipFree(d_denseDoseMat));

    checkCudaErrors(hipEventDestroy(globalStop));
    checkCudaErrors(hipEventDestroy(globalStart));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipEventDestroy(start));

    return 0;
}