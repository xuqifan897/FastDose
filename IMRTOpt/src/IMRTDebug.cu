#include "hip/hip_runtime.h"
#include <boost/filesystem.hpp>
#include "IMRTDebug.cuh"
#include "IMRTDoseMat.cuh"
#include "IMRTArgs.h"

namespace fs = boost::filesystem;
namespace fd = fastdose;

bool IMRT::doseCalcDebug(
    std::vector<BeamBundle>& beam_bundles,
    fastdose::DENSITY_d& density_d,
    fastdose::SPECTRUM_h& spectrum_h,
    fastdose::KERNEL_h& kernel_h,
    hipStream_t stream
) {
    int beamIdx = getarg<int>("beamIdxDebug");
    BeamBundle& beam_bundle = beam_bundles[beamIdx];
    int nBeamlets = beam_bundle.beams_h.size();
    std::vector<fd::BEAM_d> beamlets(nBeamlets);
    for (int i=0; i<nBeamlets; i++)
        fd::beam_h2d(beam_bundle.beams_h[i], beamlets[i]);

    #if false
        std::vector<fd::BEAM_h>& beamlets_h = beam_bundle.beams_h;
        for (int i=0; i<beamlets_h.size(); i++) {
            std::cout << "Beamlet " << i << std::endl;
            std::cout << beamlets_h[i] << std::endl << std::endl;
        }
        return 0;
    #endif

    // preparation
    std::vector<fd::d_BEAM_d> h_beams;
    h_beams.reserve(nBeamlets);
    for (int i=0; i<nBeamlets; i++)
        h_beams.emplace_back(fd::d_BEAM_d(beamlets[i]));
    fd::d_BEAM_d* d_beams = nullptr;
    checkCudaErrors(hipMalloc((void**)&d_beams, nBeamlets*sizeof(fd::d_BEAM_d)));
    checkCudaErrors(hipMemcpy(d_beams, h_beams.data(),
        nBeamlets*sizeof(fd::d_BEAM_d), hipMemcpyHostToDevice));

    std::vector<float*> h_fluence_array(nBeamlets, nullptr);
    for (int i=0; i<nBeamlets; i++)
        h_fluence_array[i] = beamlets[i].fluence;
    float** d_fluence_array = nullptr;
    checkCudaErrors(hipMalloc((void***)&d_fluence_array, nBeamlets*sizeof(float*)));
    checkCudaErrors(hipMemcpy(d_fluence_array, h_fluence_array.data(),
        nBeamlets*sizeof(float*), hipMemcpyHostToDevice));

    std::vector<float*> h_TermaBEV_array(nBeamlets, nullptr);
    for (int i=0; i<nBeamlets; i++)
        h_TermaBEV_array[i] = beamlets[i].TermaBEV;
    float** d_TermaBEV_array = nullptr;
    checkCudaErrors(hipMalloc((void***)&d_TermaBEV_array, nBeamlets*sizeof(float*)));
    checkCudaErrors(hipMemcpy(d_TermaBEV_array, h_TermaBEV_array.data(),
        nBeamlets*sizeof(float*), hipMemcpyHostToDevice));

    std::vector<float*> h_DensityBEV_array(nBeamlets, nullptr);
    for (int i=0; i<nBeamlets; i++)
        h_DensityBEV_array[i] = beamlets[i].DensityBEV;
    float** d_DensityBEV_array = nullptr;
    checkCudaErrors(hipMalloc((void***)&d_DensityBEV_array, nBeamlets*sizeof(float*)));
    checkCudaErrors(hipMemcpy(d_DensityBEV_array, h_DensityBEV_array.data(),
        nBeamlets*sizeof(float*), hipMemcpyHostToDevice));

    std::vector<float*> h_DoseBEV_array(nBeamlets, nullptr);
    for (int i=0; i<nBeamlets; i++)
        h_DoseBEV_array[i] = beamlets[i].DoseBEV;
    float** d_DoseBEV_array = nullptr;
    checkCudaErrors(hipMalloc((void***)&d_DoseBEV_array, nBeamlets*sizeof(float*)));
    checkCudaErrors(hipMemcpy(d_DoseBEV_array, h_DoseBEV_array.data(),
        nBeamlets*sizeof(float*), hipMemcpyHostToDevice));

    size_t fmap_npixels = beamlets[0].fmap_size.x * beamlets[0].fmap_size.y;

    // calculate Terma collectively
    fd::TermaComputeCollective(
        fmap_npixels,
        nBeamlets,
        d_beams,
        d_fluence_array,
        d_TermaBEV_array,
        d_DensityBEV_array,
        density_d,
        spectrum_h,
        stream
    );
    hipDeviceSynchronize();
    std::cout << "Collective Terma calculation finished." << std::endl << std::endl;

    // log out data
    fs::path resultFolder(getarg<std::string>("outputFolder"));
    resultFolder /= std::string("doseCompDebug");
    if (! fs::is_directory(resultFolder))
        fs::create_directory(resultFolder);

    #if false
        // log out Terma data
        for (int i=0; i<nBeamlets; i++) {
            const fd::BEAM_d& beamlet = beamlets[i];
            size_t DoseBEVSize = beamlet.DensityBEV_pitch / sizeof(float) * beamlet.long_dim;
            std::vector<float> h_TermaBEV(DoseBEVSize, 0.0f);
            checkCudaErrors(hipMemcpy(h_TermaBEV.data(), beamlet.TermaBEV,
                DoseBEVSize*sizeof(float), hipMemcpyDeviceToHost));
            
            fs::path file = resultFolder / (std::string("BEVTerma")
                + std::to_string(i) + ".bin");
            std::ofstream f(file.string());
            if (! f.is_open()) {
                std::cerr << "Could not open file: " << file.string();
                return 1;
            }
            f.write((char*)(h_TermaBEV.data()), DoseBEVSize*sizeof(float));
            f.close();
        }

        // log out Density data
        for (int i=0; i<nBeamlets; i++) {
            const fd::BEAM_d& beamlet = beamlets[i];
            size_t DoseBEVSize = beamlet.DensityBEV_pitch / sizeof(float) * beamlet.long_dim;
            std::vector<float> h_DensityBEV(DoseBEVSize, 0.0f);
            checkCudaErrors(hipMemcpy(h_DensityBEV.data(), beamlet.DensityBEV,
                DoseBEVSize*sizeof(float), hipMemcpyDeviceToHost));
            
            fs::path file = resultFolder / (std::string("BEVDensity")
                + std::to_string(i) + ".bin");
            std::ofstream f(file.string());
            if (! f.is_open()) {
                std::cerr << "Could not open file: " << file.string();
                return 1;
            }
            f.write((char*)(h_DensityBEV.data()), DoseBEVSize*sizeof(float));
            f.close();
        }
        return 0;
    #endif

    // print the longitudinal dimensions of beamlets
    for (int i=0; i<nBeamlets; i++)
        std::cout << "Beamlet " << i << ", long_dim: "
        << beamlets[i].long_dim << std::endl;
    std::cout << std::endl;

    #if true
        // calculate Dose collectively
        fd::DoseComputeCollective(
            fmap_npixels,
            nBeamlets,
            d_beams,
            d_TermaBEV_array,
            d_DensityBEV_array,
            d_DoseBEV_array,
            kernel_h.nTheta,
            kernel_h.nPhi,
            stream
        );
        hipDeviceSynchronize();
        std::cout << "Collective dose calculation finished." << std::endl;
    #else
        for (int i=0; i<nBeamlets; i++) {
            fd::DoseComputeCollective(
                fmap_npixels,
                1,
                d_beams + i,
                d_TermaBEV_array + i,
                d_DensityBEV_array + i,
                d_DoseBEV_array + i,
                kernel_h.nTheta,
                kernel_h.nPhi,
                stream
            );
            hipDeviceSynchronize();
            std::cout << "Dose calculation. Beamlet: " << i
                << " / " << nBeamlets << std::endl;
        }
        
        for (int i=0; i<nBeamlets; i++) {
            const fd::BEAM_d& beamlet = beamlets[i];
            size_t DoseBEVSize = beamlet.DensityBEV_pitch / sizeof(float) * beamlet.long_dim;
            std::vector<float> h_DoseBEV(DoseBEVSize, 0.0f);
            checkCudaErrors(hipMemcpy(h_DoseBEV.data(), beamlet.DoseBEV,
                DoseBEVSize*sizeof(float), hipMemcpyDeviceToHost));
            
            fs::path file = resultFolder / (std::string("BEVDose")
                + std::to_string(i) + ".bin");
            std::ofstream f(file.string());
            if (! f.is_open()) {
                std::cerr << "Could not open file: " << file.string();
                return 1;
            }
            f.write((char*)(h_DoseBEV.data()), DoseBEVSize*sizeof(float));
            f.close();
        }
    #endif

    // clean-up
    checkCudaErrors(hipFree(d_DoseBEV_array));
    checkCudaErrors(hipFree(d_DensityBEV_array));
    checkCudaErrors(hipFree(d_TermaBEV_array));
    checkCudaErrors(hipFree(d_fluence_array));
    checkCudaErrors(hipFree(d_beams));

    return 0;
}


bool IMRT::sparseValidation(const MatCSREnsemble* matEns) {
    // do beam-wise dose calculation, the sum of dose constributions from all beamlets
    // get the maximum number of rows (beamlets) of all beams
    int maxNumBeamletsPerBatch = 0;
    for (int i=0; i<matEns->numMatrices; i++)
        maxNumBeamletsPerBatch = max(maxNumBeamletsPerBatch, (int)(matEns->numRowsPerMat[i]));

    // allocate the column vector, representing the weights of all beamlets
    float* d_beamletWeights = nullptr;
    checkCudaErrors(hipMalloc((void**)&d_beamletWeights,
        maxNumBeamletsPerBatch*sizeof(float)));
    std::vector<float> h_beamletWeights(maxNumBeamletsPerBatch, 1.0f);
    checkCudaErrors(hipMemcpy(d_beamletWeights, h_beamletWeights.data(),
        maxNumBeamletsPerBatch*sizeof(float), hipMemcpyHostToDevice));
    
    // allocate the output vector
    hipsparseDnVecDescr_t vecOutput = nullptr;
    float* d_vecOutput = nullptr;
    size_t numColsPerMat = matEns->numColsPerMat;
    checkCudaErrors(hipMalloc((void**)&d_vecOutput, numColsPerMat*sizeof(float)));
    checkCusparse(hipsparseCreateDnVec(&vecOutput, numColsPerMat, d_vecOutput, HIP_R_32F));
    std::vector<float> h_vecOutput(numColsPerMat, 0.0f);

    size_t bufferSize = 0;
    void* dBuffer = nullptr;

    fs::path resultFolder = fs::path(getarg<std::string>("outputFolder"));
    resultFolder /= std::string("BeamDoseMat");
    if (! fs::is_directory(resultFolder))
        fs::create_directory(resultFolder);
    
    hipsparseHandle_t handle = nullptr;
    checkCusparse(hipsparseCreate(&handle));
    for (int i=0; i<matEns->numMatrices; i++) {
        int numRows = matEns->numRowsPerMat[i];
        int numNonZero = matEns->NonZeroElements[i];
        size_t* d_csr_offsets = matEns->d_offsetsBuffer + matEns->OffsetBufferIdx[i];
        size_t* d_csr_columns = matEns->d_columnsBuffer;
        float* d_csr_values = matEns->d_valuesBuffer;
        if (i > 0) {
            d_csr_columns += matEns->CumuNonZeroElements[i-1];
            d_csr_values += matEns->CumuNonZeroElements[i-1];
        }

        hipsparseSpMatDescr_t matSparse;
        checkCusparse(hipsparseCreateCsr(
            &matSparse, numRows, numColsPerMat, numNonZero,
            d_csr_offsets, d_csr_columns, d_csr_values,
            HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_64I,
            HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
        
        // construct input vector
        hipsparseDnVecDescr_t vecInput = nullptr;
        checkCusparse(hipsparseCreateDnVec(
            &vecInput, numRows, d_beamletWeights, HIP_R_32F));

        // determine the size of an external buffer
        size_t bufferSizeLocal = 0;
        float alpha = 1;
        float beta = 0;
        checkCusparse(hipsparseSpMV_bufferSize (
            handle, HIPSPARSE_OPERATION_TRANSPOSE,
            &alpha, matSparse, vecInput, &beta, vecOutput, HIP_R_32F,
            HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSizeLocal));
        
        if (bufferSizeLocal > bufferSize) {
            std::cout << "Enlarge buffer size, from " << bufferSize
                << " to " << bufferSizeLocal << std::endl;
            bufferSize = bufferSizeLocal;
            if (dBuffer != nullptr) {
                checkCudaErrors(hipFree(dBuffer));
            }
            checkCudaErrors(hipMalloc((void**)&dBuffer, bufferSize));
        }

        checkCusparse(hipsparseSpMV(handle, HIPSPARSE_OPERATION_TRANSPOSE,
            &alpha, matSparse, vecInput, &beta, vecOutput, HIP_R_32F,
            HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));
        
        checkCusparse(hipsparseDestroyDnVec(vecInput));
        checkCusparse(hipsparseDestroySpMat(matSparse));

        // log out
        checkCudaErrors(hipMemcpyAsync(h_vecOutput.data(), d_vecOutput,
            numColsPerMat*sizeof(float), hipMemcpyDeviceToHost));
        fs::path file = resultFolder / (std::string("beam")
            + std::to_string(i) + std::string(".bin"));
        std::ofstream f_handle(file.string());
        if (! f_handle.is_open()) {
            std::cerr << "Cannot open file: " << file << std::endl;
            return 1;
        }
        f_handle.write((char*)(h_vecOutput.data()), numColsPerMat*sizeof(float));
        f_handle.close();
        std::cout << file << std::endl;
    }

    if (bufferSize > 0)
        checkCudaErrors(hipFree(dBuffer));
    checkCusparse(hipsparseDestroyDnVec(vecOutput));
    checkCudaErrors(hipFree(d_vecOutput));
    if (maxNumBeamletsPerBatch > 0)
        checkCudaErrors(hipFree(d_beamletWeights));
    return 0;
}


bool IMRT::conversionValidation(
    const MatCSR& mat, const MatCSREnsemble& matEns
) {
    const std::vector<size_t>& numRowsPerMat = matEns.numRowsPerMat;
    size_t numRowsTotal = matEns.CumuNumRowsPerMat.back();
    std::vector<float> h_beamletWeights(numRowsTotal, 0);
    float* d_beamletWeights = nullptr;
    checkCudaErrors(hipMalloc((void**)&d_beamletWeights, numRowsTotal*sizeof(float)));

    // allocate buffer
}