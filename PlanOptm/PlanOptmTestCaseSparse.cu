#include "hip/hip_runtime.h"
#include <hipsparse.h>
#include "hip/hip_runtime_api.h"

#include "PlanOptmTestCase.cuh"

namespace fd = fastdose;

bool PlanOptm::beamBundleTestCaseSparse(
    std::vector<BeamBundle> beam_bundles,
    fastdose::DENSITY_h& density_h,
    fastdose::DENSITY_d& density_d,
    fastdose::SPECTRUM_h& spectrum_h,
    fastdose::KERNEL_h& kernel_h,
    hipStream_t stream
) {
    BeamBundle& first_beam_bundle = beam_bundles[0];
    int nBeamlets = first_beam_bundle.fluenceDim.x * first_beam_bundle.fluenceDim.y;
    first_beam_bundle.beams_d.resize(nBeamlets);
    for (int i=0; i<nBeamlets; i++) {
        fd::beam_h2d(first_beam_bundle.beams_h[i], first_beam_bundle.beams_d[i]);
    }

    // preparation
    std::vector<fd::d_BEAM_d> h_beams;
    h_beams.reserve(nBeamlets);
    for (int i=0; i<nBeamlets; i++)
        h_beams.push_back(fd::d_BEAM_d(first_beam_bundle.beams_d[i]));
    fd::d_BEAM_d* d_beams = nullptr;
    checkCudaErrors(hipMalloc((void**)&d_beams, nBeamlets*sizeof(fd::d_BEAM_d)));
    checkCudaErrors(hipMemcpy(d_beams, h_beams.data(),
        nBeamlets*sizeof(fd::d_BEAM_d), hipMemcpyHostToDevice));
    
    // allocate fluence array
    std::vector<float*> h_fluence_array(nBeamlets, nullptr);
    for (int i=0; i<nBeamlets; i++)
        h_fluence_array[i] = first_beam_bundle.beams_d[i].fluence;
    float** fluence_array = nullptr;
    checkCudaErrors(hipMalloc((void***)&fluence_array, nBeamlets*sizeof(float*)));
    checkCudaErrors(hipMemcpy(fluence_array, h_fluence_array.data(),
        nBeamlets*sizeof(float*), hipMemcpyHostToDevice));
    
    // allocate Terma_array
    std::vector<float*> h_TermaBEV_array(nBeamlets, nullptr);
    for (int i=0; i<nBeamlets; i++)
        h_TermaBEV_array[i] = first_beam_bundle.beams_d[i].TermaBEV;
    float** TermaBEV_array = nullptr;
    checkCudaErrors(hipMalloc((void***)&TermaBEV_array, nBeamlets*sizeof(float*)));
    checkCudaErrors(hipMemcpy(TermaBEV_array, h_TermaBEV_array.data(),
        nBeamlets*sizeof(float*), hipMemcpyHostToDevice));
    
    // allocate DenseBEV_array
    std::vector<float*> h_DensityBEV_array(nBeamlets, nullptr);
    for (int i=0; i<nBeamlets; i++)
        h_DensityBEV_array[i] = first_beam_bundle.beams_d[i].DensityBEV;
    float** DensityBEV_array = nullptr;
    checkCudaErrors(hipMalloc((void***)&DensityBEV_array, nBeamlets*sizeof(float*)));
    checkCudaErrors(hipMemcpy(DensityBEV_array, h_DensityBEV_array.data(),
        nBeamlets*sizeof(float*), hipMemcpyHostToDevice));

    // allocate DoseBEV_array
    std::vector<float*> h_DoseBEV_array(nBeamlets, nullptr);
    for (int i=0; i<nBeamlets; i++)
        h_DoseBEV_array[i] = first_beam_bundle.beams_d[i].DoseBEV;
    float** DoseBEV_array = nullptr;
    checkCudaErrors(hipMalloc((void***)&DoseBEV_array, nBeamlets*sizeof(float*)));
    checkCudaErrors(hipMemcpy(DoseBEV_array, h_DoseBEV_array.data(),
        nBeamlets*sizeof(float*), hipMemcpyHostToDevice));
    
    size_t fmap_npixels = first_beam_bundle.subFluenceDim.x *
        first_beam_bundle.subFluenceDim.y;

    // for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    fd::TermaComputeCollective(
        fmap_npixels,
        nBeamlets,
        d_beams,
        fluence_array,
        TermaBEV_array,
        DensityBEV_array,
        density_d,
        spectrum_h,
        stream
    );

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Terma time elapsed: " << milliseconds << " [ms]" << std::endl;


    hipEventRecord(start);

    fd::DoseComputeCollective(
        fmap_npixels,
        nBeamlets,
        d_beams,
        TermaBEV_array,
        DensityBEV_array,
        DoseBEV_array,
        kernel_h.nTheta,
        kernel_h.nPhi,
        stream
    );

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Dose time elapsed: " << milliseconds << " [ms]" << std::endl;

    // destination array
    // the density matrices are used to construct the sparse matrices.
    float* DosePVCSCollective = nullptr;
    size_t pitch_in_elements = ((density_d.VolumeDim.x + pitchModule - 1) 
        / pitchModule) * pitchModule;
    size_t singleMatrixSize = pitch_in_elements * density_d.VolumeDim.y * density_d.VolumeDim.z;
    size_t totalMatrixSize = singleMatrixSize * nBeamlets;
    checkCudaErrors(hipMalloc((void**)&DosePVCSCollective, totalMatrixSize*sizeof(float)));
    checkCudaErrors(hipMemset(DosePVCSCollective, 0.f, totalMatrixSize*sizeof(float)));

    for (int i=0; i<nBeamlets; i++) {
        fd::BEAM_d& current_beamlet = first_beam_bundle.beams_d[i];

        // for safety check
        if (current_beamlet.fmap_size.x * current_beamlet.fmap_size.y
            != current_beamlet.DoseBEV_pitch / sizeof(float)) {
            std::cerr << "The DoseBEV pitch value " << current_beamlet.DoseBEV_pitch / sizeof(float)
                << " does not equal to the size of the fluence: " << current_beamlet.fmap_size
                << std::endl;
            return 1;
        }

        hipArray* DoseBEV_Arr;
        hipTextureObject_t DoseBEV_Tex;
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
        hipExtent volumeSize = make_hipExtent(current_beamlet.fmap_size.x, 
            current_beamlet.fmap_size.y, current_beamlet.long_dim);
        hipMalloc3DArray(&DoseBEV_Arr, &channelDesc, volumeSize);
        // copy to hipArray
        hipMemcpy3DParms copyParams = {0};
        copyParams.srcPtr = make_hipPitchedPtr(
            (void*)(current_beamlet.DoseBEV),
            volumeSize.width*sizeof(float),
            volumeSize.width,
            volumeSize.height);
        copyParams.dstArray = DoseBEV_Arr;
        copyParams.extent = volumeSize;
        copyParams.kind = hipMemcpyDeviceToDevice;
        checkCudaErrors(hipMemcpy3D(&copyParams));

        hipResourceDesc texRes;
        memset(&texRes, 0, sizeof(hipResourceDesc));
        texRes.resType = hipResourceTypeArray;
        texRes.res.array.array = DoseBEV_Arr;

        hipTextureDesc texDescr;
        memset(&texDescr, 0, sizeof(hipTextureDesc));
        texDescr.normalizedCoords = false;
        texDescr.addressMode[0] = hipAddressModeBorder;
        texDescr.addressMode[1] = hipAddressModeBorder;
        texDescr.addressMode[2] = hipAddressModeBorder;
        checkCudaErrors(hipCreateTextureObject(&DoseBEV_Tex, &texRes, &texDescr, NULL));

        // prepare DosePVCS_Arr
        hipPitchedPtr DosePVCS_Arr;
        DosePVCS_Arr.ptr = DosePVCSCollective + i * singleMatrixSize;
        DosePVCS_Arr.pitch = pitch_in_elements * sizeof(float);

        fd::BEV2PVCS_SuperSampling(current_beamlet,
            density_d, DosePVCS_Arr, DoseBEV_Tex, 5, 2.0f, stream);

        // clean up
        checkCudaErrors(hipDestroyTextureObject(DoseBEV_Tex));
        checkCudaErrors(hipFreeArray(DoseBEV_Arr));
    }

    #if false
        // non-transpose
        // sparsification
        hipsparseHandle_t handle = nullptr;
        hipsparseSpMatDescr_t matSparse;
        hipsparseDnMatDescr_t matDense;
        void* dBuffer = nullptr;
        size_t bufferSize = 0;

        // Device memory management
        int* d_csr_offsets;
        int* d_csr_columns;
        float* d_csr_values;

        CHECK_CUDA(hipMalloc((void**)&d_csr_offsets,(nBeamlets+1)*sizeof(int)))
        CHECK_CUSPARSE(hipsparseCreate(&handle))
        // create dense matrix:
        CHECK_CUSPARSE(hipsparseCreateDnMat(
            &matDense, nBeamlets, singleMatrixSize, singleMatrixSize,
            DosePVCSCollective, HIP_R_32F, HIPSPARSE_ORDER_ROW))
        // create sparse matrix;
        CHECK_CUSPARSE(hipsparseCreateCsr(
            &matSparse, nBeamlets, singleMatrixSize, 0,
            d_csr_offsets, nullptr, nullptr,
            HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F))
        // allocate an external buffer if needed
        CHECK_CUSPARSE(hipsparseDenseToSparse_bufferSize(
            handle, matDense, matSparse,
            HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
            &bufferSize));
        CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))
        // execute dense to sparse conversion
        CHECK_CUSPARSE(hipsparseDenseToSparse_analysis(
            handle, matDense, matSparse,
            HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
            dBuffer))
        int64_t num_rows_tmp, num_cols_tmp, nnz;
        CHECK_CUSPARSE(hipsparseSpMatGetSize(
            matSparse, &num_rows_tmp, &num_cols_tmp, &nnz))
        // allocate CSF column indices and values
        CHECK_CUDA(hipMalloc((void**)&d_csr_columns, nnz*sizeof(int)))
        CHECK_CUDA(hipMalloc((void**)&d_csr_values, nnz*sizeof(float)));
        // reset offsets, column indices, and value pointers
        CHECK_CUSPARSE(hipsparseCsrSetPointers(
            matSparse, d_csr_offsets, d_csr_columns, d_csr_values))
        CHECK_CUSPARSE(hipsparseDenseToSparse_convert(
            handle, matDense, matSparse,
            HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
            dBuffer))
        CHECK_CUDA(hipFree(dBuffer))
        std::cout << "Number of non-zero elements: " << nnz << std::endl;


        #if false
            // sparse to dense test
            float* d_dense_ref;
            checkCudaErrors(hipMalloc((void**)&d_dense_ref,
                totalMatrixSize*sizeof(float)));
            hipsparseDnMatDescr_t matDenseRef;
            CHECK_CUSPARSE(hipsparseCreateDnMat(
                &matDenseRef, nBeamlets, singleMatrixSize, singleMatrixSize,
                d_dense_ref, HIP_R_32F, HIPSPARSE_ORDER_ROW))
            CHECK_CUSPARSE(hipsparseSparseToDense_bufferSize(
                handle, matSparse, matDenseRef,
                HIPSPARSE_SPARSETODENSE_ALG_DEFAULT,
                &bufferSize))
            CHECK_CUDA(hipMalloc((void**)&dBuffer, bufferSize))
            CHECK_CUSPARSE(hipsparseSparseToDense(
                handle, matSparse, matDenseRef,
                HIPSPARSE_SPARSETODENSE_ALG_DEFAULT,
                dBuffer))
            // compare results
            std::vector<float> h_dense_ref(totalMatrixSize, 0.0f);
            std::vector<float> h_DosePVCSCollective(totalMatrixSize, 0.0f);
            CHECK_CUDA(hipMemcpy(h_dense_ref.data(), d_dense_ref,
                totalMatrixSize*sizeof(float), hipMemcpyDeviceToHost))
            CHECK_CUDA(hipMemcpy(h_DosePVCSCollective.data(),
                DosePVCSCollective, totalMatrixSize*sizeof(float),
                hipMemcpyDeviceToHost))
            double diff_abs = 0.0;
            for (size_t i=0; i<totalMatrixSize; i++)
                diff_abs += std::abs(h_dense_ref[i] - h_DosePVCSCollective[i]);
            std::cout << "Absolute difference: " << diff_abs << std::endl;
            CHECK_CUSPARSE(hipsparseDestroyDnMat(matDenseRef))
            CHECK_CUDA(hipFree(d_dense_ref))
            CHECK_CUDA(hipFree(dBuffer))
        #endif


        #if true
            // matrix multiplication test
            float alpha = 1.0f;
            float beta = 1.0f;
            hipsparseDnVecDescr_t vecX, vecY;
            float* dX, *dY;
            CHECK_CUDA(hipMalloc((void**)&dX, singleMatrixSize * sizeof(float)))
            CHECK_CUDA(hipMalloc((void**)&dY, nBeamlets * sizeof(float)))
            std::vector<float> hX(singleMatrixSize, 1.0f);
            checkCudaErrors(hipMemcpy(dX, hX.data(),
                singleMatrixSize*sizeof(float), hipMemcpyHostToDevice));
            CHECK_CUDA(hipMemset(dY, 0.0f, nBeamlets*sizeof(float)))
            CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, singleMatrixSize, dX, HIP_R_32F))
            CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, nBeamlets, dY, HIP_R_32F))
            CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
                handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                &alpha, matSparse, vecX, &beta, vecY, HIP_R_32F,
                HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize))
            CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))

            hipEventRecord(start);
            CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                &alpha, matSparse, vecX, &beta, vecY, HIP_R_32F,
                HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer))
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&milliseconds, start, stop);
            std::cout << "Transpose computation time: " << milliseconds << " [ms]" << std::endl;
            
            // read result
            std::vector<float> hY(nBeamlets, 0.0f);
            checkCudaErrors(hipMemcpy(hY.data(), dY,
                nBeamlets*sizeof(float), hipMemcpyDeviceToHost));

            // calculate reference value
            std::vector<float> h_DosePVCSCollective(totalMatrixSize, 0.0f);
            checkCudaErrors(hipMemcpy(h_DosePVCSCollective.data(),
                DosePVCSCollective, totalMatrixSize*sizeof(float),
                hipMemcpyDeviceToHost));
            std::vector<float> hYRef(nBeamlets, 0.0f);
            for (int i=0; i<nBeamlets; i++) {
                for (int j=0; j<singleMatrixSize; j++) {
                    hYRef[i] += h_DosePVCSCollective[i*singleMatrixSize+j];
                }
            }
            float absolute_diff = 0.0f;
            double scale = 0.0;
            for (int i=0; i<nBeamlets; i++) {
                absolute_diff += std::abs(hYRef[i] - hY[i]);
                scale += hY[i];
            }
            std::cout << "Absolute difference: " << absolute_diff << 
                " / Scale: " << scale << std::endl;

            CHECK_CUDA(hipFree(dBuffer))
            CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX))
            CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY))
            CHECK_CUDA(hipFree(dX))
            CHECK_CUDA(hipFree(dY))
        #endif
        CHECK_CUSPARSE(hipsparseDestroyDnMat(matDense))
        CHECK_CUSPARSE(hipsparseDestroySpMat(matSparse))
        CHECK_CUDA(hipFree(d_csr_offsets))
        CHECK_CUDA(hipFree(d_csr_columns))
        CHECK_CUDA(hipFree(d_csr_values))
        CHECK_CUSPARSE(hipsparseDestroy(handle))

    #elif true
        // transpose
        hipsparseHandle_t handle = nullptr;
        hipsparseSpMatDescr_t matSparse;
        int* d_csr_offsets;
        CHECK_CUDA(hipMalloc((void**)&d_csr_offsets, (singleMatrixSize+1)*sizeof(float)))
        int* d_csr_columns;
        float* d_csr_values;
        hipsparseDnMatDescr_t matDense;
        void* dBuffer = nullptr;
        size_t bufferSize = 0;
        CHECK_CUSPARSE(hipsparseCreate(&handle))
        CHECK_CUSPARSE(hipsparseCreateDnMat(
            &matDense, singleMatrixSize, nBeamlets, singleMatrixSize, DosePVCSCollective,
            HIP_R_32F, HIPSPARSE_ORDER_COL))

        CHECK_CUSPARSE(hipsparseCreateCsr(
            &matSparse, singleMatrixSize, nBeamlets, 0,
            d_csr_offsets, nullptr, nullptr,
            HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F))
        
        CHECK_CUSPARSE(hipsparseDenseToSparse_bufferSize(
            handle, matDense, matSparse,
            HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
            &bufferSize))
        
        CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))

        CHECK_CUSPARSE(hipsparseDenseToSparse_analysis(
            handle, matDense, matSparse,
            HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
            dBuffer))
        
        int64_t num_rows_tmp, num_cols_tmp, nnz;
        CHECK_CUSPARSE(hipsparseSpMatGetSize(matSparse, &num_rows_tmp, &num_cols_tmp,
            &nnz))
        
        CHECK_CUDA(hipMalloc((void**)&d_csr_columns, nnz*sizeof(float)))
        CHECK_CUDA(hipMalloc((void**)&d_csr_values, nnz*sizeof(float)))
        CHECK_CUSPARSE(hipsparseCsrSetPointers(matSparse, d_csr_offsets, d_csr_columns, d_csr_values))
        CHECK_CUSPARSE(hipsparseDenseToSparse_convert(
            handle, matDense, matSparse,
            HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
            dBuffer))
        CHECK_CUDA(hipFree(dBuffer))
        
        
        // matrix-vector multiplication experiment
        hipsparseDnVecDescr_t vecX, vecY;
        float* dX;
        float* dY;
        float alpha = 1.0f;
        float beta = 1.0f;
        std::vector<float> hX(nBeamlets, 0.0f);
        CHECK_CUDA(hipMalloc((void**)&dX, nBeamlets*sizeof(float)))
        CHECK_CUDA(hipMalloc((void**)&dY, singleMatrixSize*sizeof(float)))
        CHECK_CUDA(hipMemset(dY, 0.0f, singleMatrixSize*sizeof(float)))
        CHECK_CUDA(hipMemcpy(dX, hX.data(), nBeamlets*sizeof(float), hipMemcpyHostToDevice))
        CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, nBeamlets, dX, HIP_R_32F))
        CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, singleMatrixSize, dY, HIP_R_32F))
        CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
            handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matSparse, vecX, &beta, vecY, HIP_R_32F,
            HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize))
        CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))

        hipEventRecord(start);
        CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matSparse, vecX, &beta, vecY, HIP_R_32F,
            HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer))
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        std::cout << "Number of non-zero elements: " << nnz << std::endl;
        std::cout << "Non-transpose computation time: " << milliseconds << " [ms]" << std::endl;

        CHECK_CUDA(hipFree(dBuffer))

        CHECK_CUDA(hipFree(dX))
        CHECK_CUDA(hipFree(dY))

        CHECK_CUSPARSE(hipsparseDestroyDnMat(matDense))
        CHECK_CUSPARSE(hipsparseDestroySpMat(matSparse))
        CHECK_CUSPARSE(hipsparseDestroy(handle))
        CHECK_CUDA(hipFree(d_csr_columns))
        CHECK_CUDA(hipFree(d_csr_offsets))
        CHECK_CUDA(hipFree(d_csr_values))
    #endif

    CHECK_CUDA(hipFree(DosePVCSCollective))

    // clean up
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipFree(DoseBEV_array));
    checkCudaErrors(hipFree(DensityBEV_array));
    checkCudaErrors(hipFree(TermaBEV_array));
    checkCudaErrors(hipFree(fluence_array));
    checkCudaErrors(hipFree(d_beams));

    return 0;
}